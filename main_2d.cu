#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "spread.h"
#include "utils.h"

using namespace std;

#define INFO
//#define DEBUG
//#define RESULT
#define TIME
#define OUTDRIVEN 0

#define rand01() ((FLT)rand()/RAND_MAX)
// unif[-1,1]:
#define IMA complex<FLT>(0.0,1.0)
#define randm11() (2*rand01() - (FLT)1.0)
#define crandm11() (randm11() + IMA*randm11())
#define PI (FLT)M_PI
#define M_1_2PI 0.159154943091895336
#define RESCALE(x,N,p) (p ? \
             ((x*M_1_2PI + (x<-PI ? 1.5 : (x>PI ? -0.5 : 0.5)))*N) : \
             (x<0 ? x+N : (x>N ? x-N : x)))

int cnufftspread2d_gpu(int nf1, int nf2, FLT* h_fw, int M, FLT *h_kx, 
                       FLT *h_ky, FLT *h_c, int bin_size_x, int bin_size_y)
{
  CNTime timer;
  dim3 threadsPerBlock;
  dim3 blocks;
  
  FLT tol=1e-6;
  int ns=std::ceil(-log10(tol/10.0));   // psi's support in terms of number of cells
  int es_c=4.0/(ns*ns);  
  FLT es_beta = 2.30 * (FLT)ns;

  FLT *d_c,*d_kx,*d_ky,*d_fw;
#if OUTDRIVEN
  // Parameter setting
  int numbins[2];
  int totalnupts;
  int nbin_block_x, nbin_block_y;

  int *d_binsize;
  int *d_binstartpts;
  int *d_sortidx;
  
  numbins[0] = ceil(nf1/bin_size_x)+2;
  numbins[1] = ceil(nf2/bin_size_y)+2; 
  // assume that bin_size_x > ns/2;
#ifdef INFO
  cout<<"[info  ] --> numbins (including ghost bins) = ["
      <<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif
  FLT *d_kxsorted,*d_kysorted,*d_csorted;
  int *h_binsize, *h_binstartpts, *h_sortidx; // For debug
#endif

  timer.restart();
  checkCudaErrors(hipMalloc(&d_kx,M*sizeof(FLT)));
  checkCudaErrors(hipMalloc(&d_ky,M*sizeof(FLT)));
  checkCudaErrors(hipMalloc(&d_c,2*M*sizeof(FLT)));
  checkCudaErrors(hipMalloc(&d_fw,2*nf1*nf2*sizeof(FLT)));
#if OUTDRIVEN
  checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
  checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
  checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
#endif
#ifdef TIME
  cout<<"[time  ]"<< " Allocating GPU memory " << timer.elapsedsec() <<" s"<<endl;
#endif

  timer.restart();  
  checkCudaErrors(hipMemcpy(d_kx,h_kx,M*sizeof(FLT),hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ky,h_ky,M*sizeof(FLT),hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_c,h_c,2*M*sizeof(FLT),hipMemcpyHostToDevice));
#ifdef TIME
  cout<<"[time  ]"<< " Copying memory from host to device " << timer.elapsedsec() <<" s"<<endl;
#endif

#if OUTDRIVEN
  h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
  h_sortidx     = (int*)malloc(M*sizeof(int));
  h_binstartpts = (int*)malloc((numbins[0]*numbins[1]+1)*sizeof(int));
  checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
  timer.restart();
  CalcBinSize_2d<<<64, (M+64-1)/64>>>(M,nf1,nf2,bin_size_x,bin_size_y,
                                      numbins[0],numbins[1],d_binsize,
                                      d_kx,d_ky,d_sortidx);
#ifdef TIME
  hipDeviceSynchronize();
  cout<<"[time  ]"<< " Kernel CalcBinSize_2d  takes " << timer.elapsedsec() <<" s"<<endl;
#endif
#ifdef DEBUG
  checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int), 
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),
                             hipMemcpyDeviceToHost));
  cout<<"[debug ] Before fill in the ghost bin size:"<<endl;
  for(int j=0; j<numbins[1]; j++){
    cout<<"[debug ] ";
    for(int i=0; i<numbins[0]; i++){
      if(i!=0) cout<<" ";
      cout <<"bin["<<i<<","<<j<<"] = "<<h_binsize[i+j*numbins[0]];
    }
    cout<<endl;
  }
  cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif
  timer.restart();
  threadsPerBlock.x = 16;
  threadsPerBlock.y = 16;
  blocks.x = (numbins[0]+threadsPerBlock.x-1)/threadsPerBlock.x;
  blocks.y = (numbins[1]+threadsPerBlock.y-1)/threadsPerBlock.y;  
  FillGhostBin_2d<<<blocks, threadsPerBlock>>>(bin_size_x, bin_size_y, numbins[0], 
                                               numbins[1], d_binsize);
#ifdef TIME
  hipDeviceSynchronize();
  cout<<"[time  ]"<< " Kernel FillGhostBin_2d takes " << timer.elapsedsec() <<" s"<<endl;
#endif
#ifdef DEBUG
  checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int), 
                             hipMemcpyDeviceToHost));
  cout<<"[debug ] After fill in the ghost bin size:"<<endl;
  for(int j=0; j<numbins[1]; j++){
    cout<<"[debug ] ";
    for(int i=0; i<numbins[0]; i++){
      if(i!=0) cout<<" ";
      cout <<"bin["<<i<<","<<j<<"] = "<<h_binsize[i+j*numbins[0]];
    }
    cout<<endl;
  }
  cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

  timer.restart();
  int THREADBLOCK_SIZE=256;
  int MIN_LARGE_ARRAY_SIZE=THREADBLOCK_SIZE*8;
  int arrayLength=THREADBLOCK_SIZE*8;
  if(numbins[0]*numbins[1] < MIN_LARGE_ARRAY_SIZE){ // 1024 is the maximum #threads per block 
    int arrayLength=THREADBLOCK_SIZE*4;
    int szWorkgroup = scanExclusiveShort(d_binstartpts, d_binsize, numbins[0]*numbins[1]/ arrayLength, arrayLength);
    //BinsStartPts_2d<<<1, numbins[0]*numbins[1]>>>(M,numbins[0]*numbins[1],
    //                                              d_binsize,d_binstartpts);
    //prescan<<<1, numbins[0]*numbins[1]/2, numbins[0]*numbins[1]>>>(numbins[0]*numbins[1],
    //                                                               d_binsize,
    //                                                               d_binstartpts);
  }else{
    int szWorkgroup = scanExclusiveLarge(d_binstartpts, d_binsize, numbins[0]*numbins[1]/ arrayLength, arrayLength);
    cout<<"number of bins can't fit in one block"<<endl;
    return 1;
  }
#ifdef TIME
  hipDeviceSynchronize();
  cout<<"[time  ]"<< " Kernel BinsStartPts_2d takes " << timer.elapsedsec() <<" s"<<endl;
#endif

#ifdef DEBUG
  checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int), 
                             hipMemcpyDeviceToHost));
  cout<<"[debug ] Result of scan bin_size array:"<<endl;
  for(int j=0; j<numbins[1]; j++){
    cout<<"[debug ] ";
    for(int i=0; i<numbins[0]; i++){
      if(i!=0) cout<<" ";
      cout <<"bin["<<i<<","<<j<<"] = "<<setw(2)<<h_binstartpts[i+j*numbins[0]];
    }
    cout<<endl;
  }
  cout<<"[debug ] Total number of nonuniform pts (include those in ghost bins) = "
      << setw(4)<<h_binstartpts[numbins[0]*numbins[1]]<<endl;
  cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

#if 0
  timer.restart();
  checkCudaErrors(hipMemcpy(&totalnupts,d_binstartpts+numbins[0]*numbins[1],sizeof(int), 
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipMalloc(&d_kxsorted,totalnupts*sizeof(FLT)));
  checkCudaErrors(hipMalloc(&d_kysorted,totalnupts*sizeof(FLT)));
  checkCudaErrors(hipMalloc(&d_csorted, 2*totalnupts*sizeof(FLT)));
#ifdef TIME
  hipDeviceSynchronize();
  cout<<"[time  ]"<< " Allocating GPU memory (need info of totolnupts) " << timer.elapsedsec() <<" s"<<endl;
#endif
  
  timer.restart();
  PtsRearrage_2d<<<64, (M+64-1)/64>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0], 
                                      numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted, 
                                      d_ky, d_kysorted, d_c, d_csorted);
#ifdef TIME
  hipDeviceSynchronize();
  cout<<"[time  ]"<< " Kernel PtsRearrange_2d takes " << timer.elapsedsec() <<" s"<<endl;
#endif
#ifdef DEBUG 
  FLT *h_kxsorted, *h_kysorted, *h_csorted;
  h_kxsorted = (FLT*)malloc(totalnupts*sizeof(FLT));
  h_kysorted = (FLT*)malloc(totalnupts*sizeof(FLT));
  h_csorted  = (FLT*)malloc(2*totalnupts*sizeof(FLT));
  checkCudaErrors(hipMemcpy(h_kxsorted,d_kxsorted,totalnupts*sizeof(FLT),
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_kysorted,d_kysorted,totalnupts*sizeof(FLT),
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_csorted,d_csorted,2*totalnupts*sizeof(FLT),
                             hipMemcpyDeviceToHost));
  for (int i=0; i<totalnupts; i++){
    cout <<"[debug ] (x,y) = ("<<setw(10)<<h_kxsorted[i]<<","
         <<setw(10)<<h_kysorted[i]<<"), bin# =  "
         <<(floor(h_kxsorted[i]/bin_size_x)+1)+numbins[0]*(floor(h_kysorted[i]/bin_size_y)+1)<<endl;
  }
  free(h_kysorted);
  free(h_kxsorted);
  free(h_csorted);
#endif
  
  timer.restart();
  threadsPerBlock.x = 32;
  threadsPerBlock.y = 32;
  blocks.x = (nf1 + threadsPerBlock.x - 1)/threadsPerBlock.x;
  blocks.y = (nf2 + threadsPerBlock.y - 1)/threadsPerBlock.y;
  nbin_block_x = threadsPerBlock.x/bin_size_x<(numbins[0]-2) ? threadsPerBlock.x/bin_size_x : (numbins[0]-2); 
  nbin_block_y = threadsPerBlock.y/bin_size_y<(numbins[1]-2) ? threadsPerBlock.y/bin_size_y : (numbins[1]-2); 
#ifdef INFO
  cout<<"[info  ]"<<" ["<<nf1<<"x"<<nf2<<"] "<<"output elements is divided into ["
      <<blocks.x<<","<<blocks.y<<"] block"<<", each block has ["<<nbin_block_x<<"x"<<nbin_block_y<<"] bins, "
      <<"["<<threadsPerBlock.x<<"x"<<threadsPerBlock.y<<"] threads"<<endl;
#endif
  // blockSize must be a multiple of bin_size_x 
  Spread_2d_Odriven<<<blocks, threadsPerBlock>>>(nbin_block_x, nbin_block_y, numbins[0], numbins[1], 
                                                 d_binstartpts, d_kxsorted, d_kysorted, d_csorted, 
                                                 d_fw, ns, nf1, nf2, es_c, es_beta);
#endif
#else // OUTDRIVEN
  timer.restart();
  threadsPerBlock.x = 64;
  threadsPerBlock.y = 1;
  blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
  blocks.y = 1;
  Spread_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, d_fw, M, ns,
                                                 nf1, nf2, es_c, es_beta);
#endif
#ifdef TIME
  hipDeviceSynchronize();
  cout<<"[time  ]"<< " Kernel Spread_2d takes " << timer.elapsedsec() <<" s"<<endl;
#endif
  timer.restart();
  checkCudaErrors(hipMemcpy(h_fw,d_fw,2*nf1*nf2*sizeof(FLT),
                             hipMemcpyDeviceToHost));
#ifdef TIME
  hipDeviceSynchronize();
  cout<<"[time  ]"<< " Copying memory from device to host " << timer.elapsedsec() <<" s"<<endl;
#endif
  
// Free memory
  hipFree(d_kx);
  hipFree(d_ky);
  hipFree(d_c);
  hipFree(d_fw);
#if OUTDRIVEN
  hipFree(d_binsize);
  hipFree(d_binstartpts);
  hipFree(d_sortidx);
  hipFree(d_kxsorted);
  hipFree(d_kysorted);
  hipFree(d_csorted);
  free(h_binsize); 
  free(h_binstartpts);
  free(h_sortidx);
#endif
  return 0;
}

int main(int argc, char* argv[])
{
  cout<<setprecision(3)<<endl;
  int N1 = 128, N2 = 128;
  int M = N1*N2;
  FLT sigma = 2.0;
  int bin_size_x = 16;
  int bin_size_y = 16;
  int nf1 = (int) sigma*N1;
  int nf2 = (int) sigma*N2;
  
  FLT *x, *y;
  complex<FLT> *c, *fw;
  x  = (FLT*) malloc(M*sizeof(FLT));
  y  = (FLT*) malloc(M*sizeof(FLT));
  c  = (complex<FLT>*) malloc(M*sizeof(complex<FLT>));
  fw = (complex<FLT>*) malloc(nf1*nf2*sizeof(complex<FLT>));

  for (int i = 0; i < M; i++) {
    x[i] = M_PI*randm11();// x in [-pi,pi)
    y[i] = M_PI*randm11();
    c[i] = crandm11();
  }
#ifdef INFO
  cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"<<endl;
  cout<<"[info  ] Dividing the uniform grids to bin size["<<bin_size_x<<"x"<<bin_size_y<<"]"<<endl;
#endif
  CNTime timer;
  /*warm up gpu*/
  char *a;
  timer.restart();
  checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
  cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl;
#endif

  timer.restart();
  int ier = cnufftspread2d_gpu(nf1, nf2, (FLT*) fw, M, x, y,
                               (FLT*) c, bin_size_x, bin_size_y);
  FLT ti=timer.elapsedsec();
#ifdef TIME
  printf("[info  ] %ld NU pts to (%ld,%ld) modes in %.3g s \t%.3g NU pts/s\n",M,N1,N2,ti,M/ti);
#endif
#ifdef RESULT
  cout<<"[result]"<<endl;
  for(int j=0; j<nf2; j++){
    if( j % bin_size_y == 0)
        cout<<endl;
    for (int i=0; i<nf1; i++){
      if( i % bin_size_x == 0 && i!=0)
        cout<< " |";
      //cout<<"fw[" <<i <<","<<j<<"]="<<fw[i+j*nf1];
      cout<<" "<<setw(8)<<fw[i+j*nf1];
    }
    cout<<endl;
  }
  cout<<endl;
#endif
  free(x);
  free(c);
  free(fw);
  return 0;
}

