#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "utils.h"

using namespace std;

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
static __inline__ __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

static __forceinline__ __device__
FLT evaluate_kernel(FLT x, FLT es_c, FLT es_beta)
/* ES ("exp sqrt") kernel evaluation at single real argument:
      phi(x) = exp(beta.sqrt(1 - (2x/n_s)^2)),    for |x| < nspread/2
   related to an asymptotic approximation to the Kaiser--Bessel, itself an
   approximation to prolate spheroidal wavefunction (PSWF) of order 0.
   This is the "reference implementation", used by eg common/onedim_* 2/17/17 */
{   
  //return exp(es_beta * (sqrt(1.0 - es_c*x*x) - 1));
  //return x;
  return 1.0;
}

static __inline__ __device__
void eval_kernel_vec_Horner(FLT *ker, const FLT x, const int w, const double upsampfac)
/* Fill ker[] with Horner piecewise poly approx to [-w/2,w/2] ES kernel eval at
   x_j = x + j,  for j=0,..,w-1.  Thus x in [-w/2,-w/2+1].   w is aka ns.
   This is the current evaluation method, since it's faster (except i7 w=16).
   Two upsampfacs implemented. Params must match ref formula. Barnett 4/24/18 */
{
  FLT z = 2*x + w - 1.0;         // scale so local grid offset z in [-1,1]
  // insert the auto-generated code which expects z, w args, writes to ker...
  if (upsampfac==2.0) {     // floating point equality is fine here
#include "ker_horner_allw_loop.c"
  }
}

__global__
void CalcBinSize_2d(int M, int nf1, int nf2, int  bin_size_x, int bin_size_y, int nbinx,
                    int nbiny, int* bin_size, FLT *x, FLT *y, int* sortidx)
{
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  int binidx, binx, biny;
  int oldidx;
  FLT x_rescaled,y_rescaled;
  if (i < M){
    //x_rescaled = RESCALE(x[i],nf1,1);
    //y_rescaled = RESCALE(y[i],nf2,1);
    x_rescaled=x[i];
    y_rescaled=y[i];
    binx = floor(x_rescaled/bin_size_x)+1;
    biny = floor(y_rescaled/bin_size_y)+1;
    binidx = binx+biny*nbinx;
    oldidx = atomicAdd(&bin_size[binidx], 1);
    sortidx[i] = oldidx;
  }
}

__global__
void FillGhostBin_2d(int nbinx, int nbiny, int*bin_size)
{
  int ix = blockDim.x*blockIdx.x + threadIdx.x;
  int iy = blockDim.y*blockIdx.y + threadIdx.y;
  if ( ix < nbinx && iy < nbiny){
    if(iy == 0)
      bin_size[ix+iy*nbinx] = bin_size[ix+(nbiny-2)*nbinx];
    if(iy == nbiny-1)
      bin_size[ix+iy*nbinx] = bin_size[ix+1*nbinx];
    if(ix == 0)
      bin_size[ix+iy*nbinx] = bin_size[(nbinx-2)+iy*nbinx];
    if(ix == nbinx-1)
      bin_size[ix+iy*nbinx] = bin_size[1+iy*nbinx];
    if(ix == 0 && iy == 0)
      bin_size[ix+iy*nbinx] = bin_size[(nbinx-2)+(nbiny-2)*nbinx];
    if(ix == 0 && iy == nbiny-1)
      bin_size[ix+iy*nbinx] = bin_size[(nbinx-2)+1*nbinx];
    if(ix == nbinx-1 && iy == 0)
      bin_size[ix+iy*nbinx] = bin_size[1+(nbiny-2)*nbinx];
    if(ix == nbinx-1 && iy == nbiny-1)
      bin_size[ix+iy*nbinx] = bin_size[1+1*nbinx];
  }
}

// An exclusive scan of bin_size, only works for 1 block (!)
__global__
void BinsStartPts_2d(int M, int totalnumbins, int* bin_size, int* bin_startpts)
{
  __shared__ int temp[max_shared_mem];
  int i = threadIdx.x;
  //temp[i] = (i > 0) ? bin_size[i-1] : 0;
  if ( i < totalnumbins){
    temp[i] = (i<totalnumbins) ? bin_size[i]:0;
    __syncthreads();
    for(int offset = 1; offset < totalnumbins; offset*=2){
      if( i >= offset)
        temp[i] += temp[i - offset];
      else
        temp[i] = temp[i];
      __syncthreads();
    }
    bin_startpts[i+1] = temp[i];
    if(i == 0)
      bin_startpts[i] = 0;
  }
}

__global__
void prescan(int n, int* bin_size, int* bin_startpts, int* scanblock_sum)
// only works for n is power of 2
{
  __shared__ int temp[max_shared_mem];
  int thid=threadIdx.x;
  int offset=1;
  int nelem=2*blockDim.x;

  if(2*thid+1<n){
    temp[2*thid+1]=bin_size[2*thid+1];
  }else{
    temp[2*thid+1]=0;
  }
  if(2*thid<n){
    temp[2*thid]=bin_size[2*thid];
  }else{
    temp[2*thid]=0;
  }

  for(int d = nelem>>1; d>0; d>>=1)
  {
    __syncthreads();
    if(thid<d)
    {
      int ai=offset*(2*thid+1)-1;
      int bi=offset*(2*thid+2)-1;
      temp[bi]+=temp[ai];
    }
    offset*=2;
  }

  if(thid==0) {temp[nelem-1]=0;}

  for(int d=1; d<nelem; d*=2)
  {
    offset>>=1;
    __syncthreads();
    if(thid<d)
    {
      int ai=offset*(2*thid+1)-1;
      int bi=offset*(2*thid+2)-1;

      int t=temp[ai];
      temp[ai]=temp[bi];
      temp[bi]+=t;
    }
  }
  __syncthreads();
  
  if(2*thid+1<n){
    bin_startpts[2*thid+1]=temp[2*thid+1];
  }
  if(2*thid<n){
    bin_startpts[2*thid]=temp[2*thid];
  }
  *scanblock_sum=temp[n-1]+bin_size[n-1];
}

__global__
void uniformUpdate(int n, int* data, int* buffer)
{
  __shared__ int buf;
  int pos=blockIdx.x*blockDim.x+threadIdx.x;
  if( threadIdx.x ==0){
    buf=buffer[blockIdx.x];
  }
  __syncthreads();
  if(pos<n)
    data[pos] += buf;
  if(pos==0)
    data[n] = buffer[gridDim.x];
}

__global__
void PtsRearrage_2d(int M, int nf1, int nf2, int bin_size_x, int bin_size_y, int nbinx,
                    int nbiny, int* bin_startpts, int* sortidx, FLT *x, FLT *x_sorted,
                    FLT *y, FLT *y_sorted, gpuComplex *c, gpuComplex *c_sorted)
{
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  int binx, biny;
  int binidx;
  FLT x_rescaled, y_rescaled;
  if( i < M){
    //x_rescaled = RESCALE(x[i],nf1,1);
    //y_rescaled = RESCALE(y[i],nf2,1);
    x_rescaled=x[i];
    y_rescaled=y[i];
    binx = floor(x_rescaled/bin_size_x)+1;
    biny = floor(y_rescaled/bin_size_y)+1;
    binidx = binx+biny*nbinx;

    x_sorted[bin_startpts[binidx]+sortidx[i]] = x_rescaled;
    y_sorted[bin_startpts[binidx]+sortidx[i]] = y_rescaled;
    c_sorted[bin_startpts[binidx]+sortidx[i]] = c[i];

    // four edges
    if( binx == 1 ){
      binidx = (nbinx-1)+biny*nbinx;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled + nf1;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled;
    }
    if( binx == nbinx-2 ){
      binidx = biny*nbinx;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled - nf1;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled;
    }
    if( biny == 1 ){
      binidx = binx+(nbiny-1)*nbinx;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled + nf2;
    }
    if( biny == nbiny-2 ){
      binidx = binx;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled - nf2;
    }
    // four corner
    if( binx == 1 && biny == 1){
      binidx = (nbinx-1) + (nbiny-1)*nbinx;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled + nf1;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled + nf2;
    }
    if( binx == 1 && biny == nbiny-2){
      binidx = nbinx-1;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled + nf1;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled - nf2;
    }
    if( binx == nbinx-2 && biny == 1){
      binidx = (nbiny-1)*nbinx;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled - nf1;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled + nf2;
    }
    if( binx == nbinx-2 && biny == nbiny-2){
      binidx = 0;
      x_sorted[ bin_startpts[binidx]+sortidx[i] ] = x_rescaled - nf1;
      y_sorted[ bin_startpts[binidx]+sortidx[i] ] = y_rescaled - nf2;
    }
    c_sorted[ bin_startpts[binidx]+sortidx[i] ] = c[i];
  }
}

__global__
void Spread_2d_Odriven(int nbin_block_x, int nbin_block_y, int nbinx, int nbiny, 
                       int *bin_startpts, FLT *x_sorted, FLT *y_sorted, 
                       gpuComplex *c_sorted, gpuComplex *fw, int ns,
                       int nf1, int nf2, FLT es_c, FLT es_beta, int fw_width)
{
  __shared__ FLT xshared[max_shared_mem/4];
  __shared__ FLT yshared[max_shared_mem/4];
  __shared__ gpuComplex cshared[max_shared_mem/4];

  int ix = blockDim.x*blockIdx.x+threadIdx.x;// output index, coord of the index
  int iy = blockDim.y*blockIdx.y+threadIdx.y;// output index, coord of the index
  int outidx = ix + iy*fw_width;
  int binxLo = blockIdx.x*nbin_block_x;
  int binxHi = binxLo+nbin_block_x+1 < nbinx-1 ? binxLo+nbin_block_x+1 : nbinx-1;
  int binyLo = blockIdx.y*nbin_block_y;
  int binyHi = binyLo+nbin_block_y+1 < nbiny-1 ? binyLo+nbin_block_y+1 : nbiny-1;
  int start, end, j, bx, by, bin;
  FLT disx, disy, kervalue1, kervalue2;
  //FLT tr=0.0, ti=0.0;
  gpuComplex t=make_hipDoubleComplex(0,0);
  // run through all bins
  for(by=binyLo; by<=binyHi; by++){
    //for(bx=binxLo; bx<=binxHi; bx++){
      //bin = bx+by*nbinx;
      //start = bin_startpts[bin];
      //end   = bin_startpts[bin+1];
      start = bin_startpts[binxLo+by*nbinx];
      end   = bin_startpts[binxHi+by*nbinx+1];
      for(int tid=threadIdx.x+blockDim.x*threadIdx.y; tid<end-start; tid+=blockDim.x*blockDim.y){
        xshared[tid] = x_sorted[start+tid];
        yshared[tid] = y_sorted[start+tid];
        cshared[tid] = c_sorted[start+tid];
      }
      __syncthreads();
      if( ix < nf1 && iy < nf2){
        for(j=0; j<end-start; j++){
          disx = abs(xshared[j]-ix);
          disy = abs(yshared[j]-iy);
          gpuComplex c=cshared[j];
          if( (disx < 7.0/2.0) && (disy < 7.0/2.0)){
            kervalue1 = evaluate_kernel(disx, es_c, es_beta);
            kervalue2 = evaluate_kernel(disy, es_c, es_beta);
            t.x+=kervalue1*kervalue2;
            t.y+=kervalue1*kervalue2;
            //t.x++;
            //t.y++;
            //t = hipCadd(t, make_hipDoubleComplex(c.x*kervalue, c.y*kervalue));
            //tr += cshared[2*j]*kervalue;
            //ti += cshared[2*j+1]*kervalue;
          }
        }
      }
    //}
  } 
  if( ix < nf1 && iy < nf2){
    fw[outidx]=t;
  }
    //fw[2*outidx]   = tr;
    //fw[2*outidx+1] = ti;
}

__global__
void Spread_2d_Idriven(FLT *x, FLT *y, gpuComplex *c, gpuComplex *fw, int M, const int ns,
                       int nf1, int nf2, FLT es_c, FLT es_beta, int fw_width)
{
  int xstart,ystart,xend,yend;
  int xx, yy, ix, iy;
  int outidx;
  //FLT ker1[7];
  //FLT ker2[7];
  //FLT ker1val, ker2val;
  //double sigma=2.0;

  FLT x_rescaled, y_rescaled;
  for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<M; i+=blockDim.x*gridDim.x){
    //x_rescaled = RESCALE(x[i],nf1,1);
    //y_rescaled = RESCALE(y[i],nf2,1);
    x_rescaled=x[i];
    y_rescaled=y[i];
    xstart = ceil(x_rescaled - ns/2.0);
    ystart = ceil(y_rescaled - ns/2.0);
    xend = floor(x_rescaled + ns/2.0);
    yend = floor(y_rescaled + ns/2.0);

#if 0
    FLT x1=(FLT)xstart-x_rescaled;
    FLT y1=(FLT)ystart-y_rescaled;
    eval_kernel_vec_Horner(ker1,x1,ns,sigma);
    eval_kernel_vec_Horner(ker2,y1,ns,sigma);
#endif
    for(yy=ystart; yy<=yend; yy++){
#if 0
       ker2val=ker2[yy-ystart];
#endif
       for(xx=xstart; xx<=xend; xx++){
#if 0
          ker1val=ker1[xx-xstart];
#endif
          ix = xx < 0 ? xx+nf1 : (xx>nf1-1 ? xx-nf1 : xx);
          iy = yy < 0 ? yy+nf2 : (yy>nf2-1 ? yy-nf2 : yy);
          outidx = ix+iy*fw_width;
#if 0
          FLT kervalue=ker1val*ker2val;
#endif
          FLT disx=abs(x_rescaled-xx);
          FLT disy=abs(y_rescaled-yy);
          FLT kervalue1 = evaluate_kernel(disx, es_c, es_beta);
          FLT kervalue2 = evaluate_kernel(disy, es_c, es_beta);
          atomicAdd(&fw[outidx].x, kervalue1*kervalue2);
          atomicAdd(&fw[outidx].y, kervalue1*kervalue2);
          //atomicAdd(&fw[2*outidx  ], 1.0);
          //atomicAdd(&fw[2*outidx+1], 1.0);
       }
    }

  }

}

__global__
void CreateSortIdx (int M, int nf1, int nf2, FLT *x, FLT *y, int* sortidx)
{
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  FLT x_rescaled,y_rescaled;
  if (i < M){
    //x_rescaled = RESCALE(x[i],nf1,1);
    //y_rescaled = RESCALE(y[i],nf2,1);
    x_rescaled=x[i];
    y_rescaled=y[i];
    sortidx[i] = floor(x_rescaled) + floor(y_rescaled)*nf1;
  }
}
