#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/spread.h"
#include "../src/finufft/utils.h"
#include "../src/finufft/cnufftspread.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT sigma = 2.0;
	int N1, N2, M;
	if (argc<4) {
		fprintf(stderr,"Usage: accuracy [nupts_distribute [N1 N2 [M [tol]]]]\n");
		return 1;
	}  
	int nupts_distribute;
	sscanf(argv[1],"%d",&nupts_distribute);

	double w;
	sscanf(argv[2],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	M = N1*N2;// let density always be 1
	if(argc>4){
		sscanf(argv[4],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>5){
		sscanf(argv[5],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}


	int ns=std::ceil(-log10(tol/10.0));
	spread_opts opts;
	opts.nspread=ns;
	opts.upsampfac=2.0;
	opts.ES_beta= 2.30 * (FLT)ns;
	opts.ES_c=4.0/(ns*ns);
	opts.ES_halfwidth=(FLT)ns/2;
	opts.Horner=0;
	opts.maxsubprobsize=1000;
	opts.pirange=0;
	opts.bin_sort=1;

	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y;
	CPX *c, *fwic, *fwi, *fwo, *fwh, *fws, *fwfinufft;
	hipHostMalloc(&x, M*sizeof(CPX));
	hipHostMalloc(&y, M*sizeof(CPX));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fwi,       nf1*nf2*sizeof(CPX));
	hipHostMalloc(&fwic,      nf1*nf2*sizeof(CPX));
	hipHostMalloc(&fwo,       nf1*nf2*sizeof(CPX));
	hipHostMalloc(&fwh,       nf1*nf2*sizeof(CPX));
	hipHostMalloc(&fws,       nf1*nf2*sizeof(CPX));
	hipHostMalloc(&fwfinufft, nf1*nf2*sizeof(CPX));

        switch(nupts_distribute){
                // Making data
                case 1: //uniform
                {
                        for (int i = 0; i < M; i++) {
                                x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
                                y[i] = RESCALE(M_PI*randm11(), nf2, 1);
                                c[i].real() = randm11();
                                c[i].imag() = randm11();
                        }
                }
                break;
                case 2: // concentrate on a small region
                {
                        for (int i = 0; i < M; i++) {
                                x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);// x in [-pi,pi)
                                y[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf2, 1);
                                c[i].real() = randm11();
                                c[i].imag() = randm11();
                        }
                }
                break;
        }

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl<<endl;
#endif
#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"<<endl;
#endif


	/* -------------------------------------- */
	// Method 1: Input driven without sorting //
	/* -------------------------------------- */
	timer.restart();
	opts.method=1;
	ier = cnufftspread2d_gpu(nf1, nf2, fwi, M, x, y, c, opts);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d_gpu_idriven"<<endl;
		return 0;
	}
	FLT tidriven=timer.elapsedsec();
	printf("[idriven] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			M,N1,N2,nf1*nf2,tidriven,M/tidriven);

	/* -------------------------------------- */
	// Method 2: Input driven with sorting    //
	/* -------------------------------------- */
	timer.restart();
	opts.method=2;
	opts.bin_size_x=16;
	opts.bin_size_y=16;
	ier = cnufftspread2d_gpu(nf1, nf2, fwic, M, x, y, c, opts);
	FLT ticdriven=timer.elapsedsec();
	printf("[isorted] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			M,N1,N2,nf1*nf2,ticdriven,M/ticdriven);
	
	/* -------------------------------------- */
	// Method 3: Output driven                //
	/* -------------------------------------- */
	if(nupts_distribute == 1){
		timer.restart();
		opts.method=3;
		opts.bin_size_x=4;
		opts.bin_size_y=4;
		ier = cnufftspread2d_gpu(nf1, nf2, fwo, M, x, y, c, opts);
		if(ier != 0 ){
			cout<<"error: cnufftspread2d_gpu_odriven"<<endl;
			return 0;
		}
		FLT todriven=timer.elapsedsec();
		printf("[odriven] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			M,N1,N2,nf1*nf2,todriven,M/todriven);
	}

	/* -------------------------------------- */
	// Method 4: Hybrid                       //
	/* -------------------------------------- */
	timer.restart();
	opts.method=4;
	opts.bin_size_x=32;
	opts.bin_size_y=32;
	ier = cnufftspread2d_gpu(nf1, nf2, fwh, M, x, y, c, opts);
	FLT thybrid=timer.elapsedsec();
	if(ier != 0 ){
		cout<<"error: cnufftspread2d_gpu_hybrid"<<endl;
		return 0;
	}
	printf("[hybrid ] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			M,N1,N2,nf1*nf2,thybrid,M/thybrid);

	/* -------------------------------------- */
	// Method 5: Subprob                     //
	/* -------------------------------------- */
	timer.restart();
	opts.method=5;
	opts.bin_size_x=32;
	opts.bin_size_y=32;
	ier = cnufftspread2d_gpu(nf1, nf2, fws, M, x, y, c, opts);
	FLT tsubprob=timer.elapsedsec();
	if(ier != 0 ){
		cout<<"error: cnufftspread2d_gpu_subprob"<<endl;
		return 0;
	}
	printf("[subprob ] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
			 M,N1,N2,nf1*nf2,tsubprob,M/thybrid);
	/* -------------------------------------- */
	// FINUTFFT cpu spreader                  //
	/* -------------------------------------- */
	timer.start();
	setup_spreader(opts,(FLT)tol,opts.upsampfac,opts.kerevalmeth);
	opts.pirange=0;
	opts.chkbnds=1;
	opts.spread_direction=1;
	opts.flags=0;//ker always return 1
	opts.kerevalmeth=1;
	opts.kerpad=1;
	opts.sort_threads=0;
	opts.sort=2;
	opts.debug=0;

	ier = cnufftspread(nf1,nf2,1,(FLT*) fwfinufft,M,x,y,NULL,(FLT*) c,opts);
	FLT t=timer.elapsedsec();
	if (ier!=0) {
		printf("error (ier=%d)!\n",ier);
		return ier;
	}
	printf("[finufft] %ld NU pts to (%ld,%ld) modes, #%d U pts in %.3g s \t%.3g NU pts/s\n",
		M,N1,N2,nf1*nf2,t,M/t);
		//printf("    %.3g NU pts in %.3g s \t%.3g pts/s \t%.3g spread pts/s\n",(double)M,t,M/t,pow(opts.nspread,2)*M/t);
	/* ------------------------------------------------------------------------------------------------------*/
	
	cout<<endl;
	FLT err=relerrtwonorm(nf1*nf2,fwi,fwfinufft);
	printf("|| fwi  - fwfinufft ||_2 / || fwi  ||_2 =  %.6g\n", err);
	err=relerrtwonorm(nf1*nf2,fwic,fwfinufft);
	printf("|| fwic - fwfinufft ||_2 / || fwic ||_2 =  %.6g\n", err);
	if(nupts_distribute == 1){
		err=relerrtwonorm(nf1*nf2,fwo,fwfinufft);
		printf("|| fwo  - fwfinufft ||_2 / || fwo  ||_2 =  %.6g\n", err);
	}
	err=relerrtwonorm(nf1*nf2,fwh,fwfinufft);
	printf("|| fwh  - fwfinufft ||_2 / || fwh  ||_2 =  %.6g\n", err);
	err=relerrtwonorm(nf1*nf2,fws,fwfinufft);
	printf("|| fws  - fwfinufft ||_2 / || fwh  ||_2 =  %.6g\n", err);

#ifdef RESULT
	cout<<"[resultdiff]"<<endl;
	FLT fwi_infnorm=infnorm(nf1*nf2, fwi);
	int nn=0;
	for(int j=0; j<nf2; j++){
		for (int i=0; i<nf1; i++){
			if( norm(fwi[i+j*nf1]-fwh[i+j*nf1])/fwi_infnorm > 1e-5 & nn<10){
				cout<<norm(fwi[i+j*nf1]-fwh[i+j*nf1])/fwi_infnorm<<" ";
				cout<<"(i,j)=("<<i<<","<<j<<"), "<<fwi[i+j*nf1] <<","<<fwh[i+j*nf1]<<endl;
				nn++;
			}
		}
	}
	cout<<endl;
#endif
#ifdef RESULT
	cout<<"[result-hybrid]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % opts.bin_size_y == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % opts.bin_size_x == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fwh[i+j*nf1].real(),fwh[i+j*nf1].imag() );
			//cout<<" "<<setw(8)<<fwo[i+j*nf1];
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fwi);
	hipHostFree(fwic);
	//hipHostFree(fwo);
	hipHostFree(fwh);
	hipHostFree(fwfinufft);
	return 0;
}
