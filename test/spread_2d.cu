#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <algorithm>
#include "../src/cufinufft.h"
#include "../src/spreadinterp.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT upsampfac=2.0;
	int N1, N2, M;
	if (argc<5) {
		fprintf(stderr,"Usage: spread2d [method [maxsubprob [nupts_distr [N1 N2 [rep [tol [kerevalmeth]]]]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: nupts driven\n");
		fprintf(stderr,"method 2: sub-problem\n");
		fprintf(stderr,"method 3: sub-problem with paul's idea\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int maxsubprobsize;
	sscanf(argv[2],"%d",&maxsubprobsize);
	int nupts_distribute;
	sscanf(argv[3],"%d",&nupts_distribute);
	sscanf(argv[4],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/upsampfac;
	N2 = (int) nf2/upsampfac;
	int rep = 10;
	if(argc>6){
		//sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		sscanf(argv[6],"%d",&rep);
		//if(M == 0) M=N1*N2*4*rep;
	}
	M = N1*N2*4*rep;// let density always be 1
	M = nf1*nf2*rep;// let density always be 1

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int kerevalmeth=0;
	if(argc>8){
		sscanf(argv[8],"%d",&kerevalmeth);
	}

	int ier;

	int dim=2;
	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_plan dplan;
	ier = cufinufft_default_opts(type1, dim, dplan.opts);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	dplan.opts.gpu_method=method;
	dplan.opts.upsampfac=upsampfac;
	dplan.opts.gpu_maxsubprobsize=maxsubprobsize;
	dplan.opts.gpu_kerevalmeth=kerevalmeth;

	cout<<scientific<<setprecision(3);


	FLT *x, *y;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*sizeof(CPX));

	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				for (int j=0; j<nf2; j++) {
					for (int i=0; i<nf1; i++){
						for (int k=0; k<rep; k++){
							if(k+i*rep+j*nf1*rep < M){
								x[k+i*rep+j*nf1*rep] = i;
								y[k+i*rep+j*nf1*rep] = j;
							}
						}
					}
				}
#if 0 
				srand(unsigned(1)); 
				random_shuffle (&x[0], &x[M-1]);
				srand(unsigned(1)); 
				random_shuffle (&y[0], &y[M-1]);
#endif
				for (int i = 0; i < M; i++) {
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;

		case 2:
			{
				for (int k=0; k<rep; k++){
					for (int j=0; j<nf2; j++) {
						for (int i=0; i<nf1; i++){
							if(i+j*nf1+k*nf1*nf2< M){
								x[i+j*nf1+k*nf1*nf2] = i;
								y[i+j*nf1+k*nf1*nf2] = j;
							}
						}
					}
				}
				for (int i = 0; i < M; i++) {
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		
		case 3:
			{
				for (int j=0; j<nf2; j++) {
					for (int i=0; i<nf1; i++){
						for (int k=0; k<rep; k++){
							if(k+i*rep+j*nf1*rep < M){
								x[k+i*rep+j*nf1*rep] = i;
								y[k+i*rep+j*nf1*rep] = j;
							}
						}
					}
				}
				srand(unsigned(1)); 
				random_shuffle (&x[0], &x[M-1]);
				srand(unsigned(1)); 
				random_shuffle (&y[0], &y[M-1]);
				for (int i = 0; i < M; i++) {
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		case 4: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01(), nf1, 1)/2.0 - 0.5;// x in [-pi,pi)
					y[i] = RESCALE(M_PI*rand01(), nf2, 1)/2.0 - 0.5;
					if(method == 6){
						x[i] = x[i] > nf1-0.5 ? x[i] - nf1 : x[i];
						y[i] = y[i] > nf2-0.5 ? y[i] - nf2 : y[i];// x in [-pi,pi)
					}
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		case 5:
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
					if(method == 6){
						x[i] = x[i] > nf1-0.5 ? x[i] - nf1 : x[i];
						y[i] = y[i] > nf2-0.5 ? y[i] - nf2 : y[i];// x in [-pi,pi)
					}
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		case 6:
			{
				for(int i=0; i<M; i++) {
					x[i] = 1;// x in [-pi,pi)
					y[i] = 1;
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() 
		<<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"
		<<endl;
#endif
	timer.restart();
	ier = cufinufft_spread2d(N1, N2, nf1, nf2, fw, M, x, y, c, tol, &dplan);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (\t%.3g NU pts/s)\n",
			dplan.opts.gpu_method,M,nf1*nf2,t,M/t);
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % dplan.opts.gpu_binsizey == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % dplan.opts.gpu_binsizex == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
		
