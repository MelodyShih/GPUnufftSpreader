#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include "../src/spreadinterp.h"
#include "../src/cufinufft.h"
#include "../src/profile.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, M;
	if (argc<4) {
		fprintf(stderr,"Usage: cufinufft2d2_test [method [N1 N2 [M [tol]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: input driven without sorting\n");
		fprintf(stderr,"method 5: subprob\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	M = N1*N2;// let density always be 1
	if(argc>4){
		sscanf(argv[4],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>5){
		sscanf(argv[5],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*sizeof(CPX));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
	}
	for(int i=0; i<N1*N2; i++){
		fk[i].real() = 1.0;
		fk[i].imag() = 1.0;
	}

	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("Warm Up",1);
		char *a;
		checkCudaErrors(hipMalloc(&a,1));
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);
#endif

	cufinufft_plan dplan;
	ier=cufinufft_default_opts(dplan.opts);
	dplan.opts.gpu_method=method;

	int dim = 2;
	int nmodes[3];
	int ntransf = 1;
	int ntransfcufftplan = 1;
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = 1;
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_plan",2);
		ier=cufinufft_makeplan(type2, dim, nmodes, iflag, ntransf, tol, 
			ntransfcufftplan, &dplan);
		if (ier!=0){
			printf("err: cufinufft2d_plan\n");
		}
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);
#endif
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_setNUpts",3);
		ier=cufinufft_setNUpts(M, x, y, NULL, 0, NULL, NULL, NULL, &dplan);
		if (ier!=0){
			printf("err: cufinufft_setNUpts\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d2_exec",4);
		ier=cufinufft_exec(c, fk, &dplan);
		if (ier!=0){
			printf("err: cufinufft2d2_exec\n");
		}
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);
#endif
	hipEventRecord(start);
	{
		PROFILE_CUDA_GROUP("cufinufft2d_destroy",5);
		ier=cufinufft_destroy(&dplan);
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);
#endif
#if 0
	// This must be here, since in gpu code, x, y gets modified if pirange=1
	int jt = M/2;          // check arbitrary choice of one targ pt
	CPX J = IMA*(FLT)iflag;
	CPX ct = CPX(0,0);
	int m=0;
	for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
		for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
			ct += fk[m++] * exp(J*(m1*x[jt] + m2*y[jt]));   // crude direct
	printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(c[jt]-ct)/infnorm(M,c));
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		//        if( j % opts.gpu_binsizey == 0)
		//                printf("\n");
		for (int i=0; i<nf1; i++){
			//                if( i % opts.gpu_binsizex == 0 && i!=0)
			//                        printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
#endif	
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
	return 0;
}
