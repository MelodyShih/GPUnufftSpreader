#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include "../src/spreadinterp.h"
#include "../src/cufinufft.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	FLT sigma = 2.0;
	int N1, N2, M;
	if (argc<4) {
		fprintf(stderr,"Usage: cufinufft2d2_test [method [N1 N2 [M [tol]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: input driven without sorting\n");
		fprintf(stderr,"method 5: subprob\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	M = N1*N2;// let density always be 1
	if(argc>4){
		sscanf(argv[4],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>5){
		sscanf(argv[5],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*sizeof(CPX));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
	}
	for(int i=0; i<N1*N2; i++){
		fk[i].real() = 1.0;
		fk[i].imag() = 1.0;
	}
	// This must be here, since in gpu code, x, y gets modified if pirange=1
	int jt = M/2;          // check arbitrary choice of one targ pt
	CPX J = IMA*(FLT)iflag;
	CPX ct = CPX(0,0);
	int m=0;
	for (int m2=-(N2/2); m2<=(N2-1)/2; ++m2)  // loop in correct order over F
		for (int m1=-(N1/2); m1<=(N1-1)/2; ++m1)
			ct += fk[m++] * exp(J*(m1*x[jt] + m2*y[jt]));   // crude direct

	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	char *a;
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);
#endif

	cufinufft_plan dplan;
	cufinufft_opts opts;
	ier=cufinufft_default_opts(opts,tol,sigma);
	opts.method=method;
	opts.spread_direction=2;

	hipEventRecord(start);
	ier=cufinufft2d_plan(M, N1, N2, iflag, opts, &dplan);
	if (ier!=0){
		printf("err: cufinufft2d_plan\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft2d_setptrs(x, y, opts, &dplan);
	if (ier!=0){
		printf("err: cufinufft2d_setptrs\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft setptrs:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft2d2_exec(c, fk, opts, &dplan);
	if (ier!=0){
		printf("err: cufinufft2d2_exec\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft2d_destroy(opts, &dplan);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);
	printf("[gpu   ] one targ: rel err in c[%ld] is %.3g\n",(int64_t)jt,abs(c[jt]-ct)/infnorm(M,c));
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		//        if( j % opts.bin_size_y == 0)
		//                printf("\n");
		for (int i=0; i<nf1; i++){
			//                if( i % opts.bin_size_x == 0 && i!=0)
			//                        printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
#endif	
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
	return 0;
}
