#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include "../src/spreadinterp.h"
#include "../src/cufinufft.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int N1, N2, N3, M, N;
	if (argc<4) {
		fprintf(stderr,"Usage: cufinufft3d1_test [method [N1 N2 N3 [M [tol]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: nupts driven\n");
		fprintf(stderr,"method 2: sub-problems\n");
		fprintf(stderr,"method 4: block gather\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); N3 = (int)w;  // so can read 1e6 right!
	
	M = N1*N2*N3;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y, *z;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*N3*sizeof(CPX));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
		z[i] = M_PI*randm11();
		c[i].real() = randm11();
		c[i].imag() = randm11();
	}

	hipEvent_t start, stop;
	float milliseconds = 0;
	float totaltime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	char *a;
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);
#endif

	cufinufft_plan dplan;

	ier=cufinufft_default_opts(dplan.opts);
	dplan.opts.gpu_method=method;
	dplan.opts.gpu_binsizex = 16;
	dplan.opts.gpu_binsizey = 16;
	dplan.opts.gpu_binsizez = 2;
	dplan.opts.gpu_maxsubprobsize = 4096;

	int dim = 3;
	int nmodes[3];
	int ntransf = 1;
	int ntransfcufftplan = 1;
	nmodes[0] = N1;
	nmodes[1] = N2;
	nmodes[2] = N3;
	hipEventRecord(start);
	ier=cufinufft_makeplan(type1, dim, nmodes, iflag, ntransf, tol, 
		ntransfcufftplan, &dplan);
	if (ier!=0){
		printf("err: cufinufft_makeplan\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);


	hipEventRecord(start);
	ier=cufinufft_setNUpts(M, x, y, z, 0, NULL, NULL, NULL, &dplan);
	if (ier!=0){
		printf("err: cufinufft_setNUpts\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft setNUpts:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft_exec(c, fk, &dplan);
	if (ier!=0){
		printf("err: cufinufft_exec\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft_destroy(&dplan);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);

	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (\t%.3g NU pts/s)\n",
			dplan.opts.gpu_method,M,N1*N2*N3,totaltime/1000,M/totaltime*1000);
	int nt1 = (int)(0.37*N1), nt2 = (int)(0.26*N2), nt3 = (int) (0.13*N3);  // choose some mode index to check
	CPX Ft = CPX(0,0), J = IMA*(FLT)iflag;
	for (BIGINT j=0; j<M; ++j)
		Ft += c[j] * exp(J*(nt1*x[j]+nt2*y[j]+nt3*z[j]));   // crude direct
	int it = N1/2+nt1 + N1*(N2/2+nt2) + N1*N2*(N3/2+nt3);   // index in complex F as 1d array
	N = N1*N2*N3;
	printf("[gpu   ] one mode: abs err in F[%ld,%ld,%ld] is %.3g\n",(int)nt1,
		(int)nt2, (int)nt3, (abs(Ft-fk[it])));
	printf("[gpu   ] one mode: rel err in F[%ld,%ld,%ld] is %.3g\n",(int)nt1,
		(int)nt2, (int)nt3, abs(Ft-fk[it])/infnorm(N,fk));
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		//        if( j % opts.bin_size_y == 0)
		//                printf("\n");
		for (int i=0; i<nf1; i++){
			//                if( i % opts.bin_size_x == 0 && i!=0)
			//                        printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
#endif	
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
	return 0;
}
