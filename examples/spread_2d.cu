#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <algorithm>
#include "../src/cufinufft.h"
#include "../src/spreadinterp.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT sigma = 2.0;
	int N1, N2, M;
	if (argc<5) {
		fprintf(stderr,"Usage: spread2d [method [maxsubprob [nupts_distr [N1 N2 [rep [tol [kerevalmeth]]]]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: input driven without sorting\n");
		fprintf(stderr,"method 2: input driven with sorting\n");
		fprintf(stderr,"method 4: hybrid\n");
		fprintf(stderr,"method 5: subprob\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int maxsubprobsize;
	sscanf(argv[2],"%d",&maxsubprobsize);
	int nupts_distribute;
	sscanf(argv[3],"%d",&nupts_distribute);
	sscanf(argv[4],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	int rep = 10;
	if(argc>6){
		//sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		sscanf(argv[6],"%d",&rep);
		//if(M == 0) M=N1*N2*4*rep;
	}
	M = N1*N2*4*rep;// let density always be 1
	M = nf1*nf2*rep;// let density always be 1

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int kerevalmeth=0;
	if(argc>8){
		sscanf(argv[8],"%d",&kerevalmeth);
	}

	int ier;

	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_opts opts;
	cufinufft_plan dplan;
	FLT upsampfac=2.0;

	ier = cufinufft_default_opts(opts,tol,upsampfac);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	opts.gpu_method=method;
	cout<<scientific<<setprecision(3);


	FLT *x, *y;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*sizeof(CPX));

	opts.pirange=0;
	opts.kerevalmeth=kerevalmeth;
	if(method == 6)
		opts.gpu_maxsubprobsize=maxsubprobsize;
	if(method == 5)
		opts.gpu_maxsubprobsize=maxsubprobsize;
	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				for (int j=0; j<nf2; j++) {
					for (int i=0; i<nf1; i++){
						for (int k=0; k<rep; k++){
							if(k+i*rep+j*nf1*rep < M){
								x[k+i*rep+j*nf1*rep] = i;
								y[k+i*rep+j*nf1*rep] = j;
							}
						}
					}
				}
#if 0 
				srand(unsigned(1)); 
				random_shuffle (&x[0], &x[M-1]);
				srand(unsigned(1)); 
				random_shuffle (&y[0], &y[M-1]);
#endif
				for (int i = 0; i < M; i++) {
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;

		case 2:
			{
				for (int k=0; k<rep; k++){
					for (int j=0; j<nf2; j++) {
						for (int i=0; i<nf1; i++){
							if(i+j*nf1+k*nf1*nf2< M){
								x[i+j*nf1+k*nf1*nf2] = i;
								y[i+j*nf1+k*nf1*nf2] = j;
							}
						}
					}
				}
				for (int i = 0; i < M; i++) {
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		
		case 3:
			{
				for (int j=0; j<nf2; j++) {
					for (int i=0; i<nf1; i++){
						for (int k=0; k<rep; k++){
							if(k+i*rep+j*nf1*rep < M){
								x[k+i*rep+j*nf1*rep] = i;
								y[k+i*rep+j*nf1*rep] = j;
							}
						}
					}
				}
				srand(unsigned(1)); 
				random_shuffle (&x[0], &x[M-1]);
				srand(unsigned(1)); 
				random_shuffle (&y[0], &y[M-1]);
				for (int i = 0; i < M; i++) {
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		case 4: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01(), nf1, 1)/2.0 - 0.5;// x in [-pi,pi)
					y[i] = RESCALE(M_PI*rand01(), nf2, 1)/2.0 - 0.5;
					if(method == 6){
						x[i] = x[i] > nf1-0.5 ? x[i] - nf1 : x[i];
						y[i] = y[i] > nf2-0.5 ? y[i] - nf2 : y[i];// x in [-pi,pi)
					}
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		case 5:
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
					if(method == 6){
						x[i] = x[i] > nf1-0.5 ? x[i] - nf1 : x[i];
						y[i] = y[i] > nf2-0.5 ? y[i] - nf2 : y[i];// x in [-pi,pi)
					}
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		case 6:
			{
				for(int i=0; i<M; i++) {
					x[i] = 1;// x in [-pi,pi)
					y[i] = 1;
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() 
		<<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"
		<<endl;
#endif
	if(opts.gpu_method == 2)
	{
		opts.gpu_binsizex=16;
		opts.gpu_binsizey=16;
	}

	if(opts.gpu_method == 4 || opts.gpu_method==5)
	{
		opts.gpu_binsizex=32;
		opts.gpu_binsizey=32;
	}

	if(opts.gpu_method == 6)
	{
		opts.gpu_binsizex=32;
		opts.gpu_binsizey=32;
	}
	timer.restart();
	ier = cufinufft_spread2d(N1, N2, nf1, nf2, fw, M, x, y, c, opts, &dplan);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (\t%.3g NU pts/s)\n",
			opts.gpu_method,M,nf1*nf2,t,M/t);
#if 0
	switch(method)
	{
		case 4:
			opts.gpu_binsizex=32;
			opts.gpu_binsizey=32;
		case 5:
			opts.gpu_binsizex=16;
			opts.gpu_binsizey=16;
		default:
			opts.gpu_binsizex=nf1;
			opts.gpu_binsizey=nf2;		
	}
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % opts.gpu_binsizey == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % opts.gpu_binsizex == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
		
