#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/spread.h"
#include "../src/finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT sigma = 2.0;
	int N1, N2, M;
	if (argc<4) {
		fprintf(stderr,"Usage: spread2d [method [nupts_distr [N1 N2 [M [tol [use_thrust]]]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: input driven without sorting\n");
		fprintf(stderr,"method 2: input driven with sorting\n");
		fprintf(stderr,"method 3: output driven\n");
		fprintf(stderr,"method t: hybrid\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	M = N1*N2;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int use_thrust=0;
	if(argc>7){
		sscanf(argv[7],"%d",&use_thrust);
	}

	int ns=std::ceil(-log10(tol/10.0));
	spread_opts opts;
	opts.nspread=ns;
	opts.upsampfac=2.0;
	opts.ES_beta=2.30*(FLT)ns;
	opts.ES_c=4.0/(ns*ns);
	opts.ES_halfwidth=(FLT)ns/2;
	opts.use_thrust=use_thrust;
	opts.method=method;
	opts.Horner=0;
	opts.pirange=0;

	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(CPX));
	hipHostMalloc(&y, M*sizeof(CPX));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*sizeof(CPX));

        switch(nupts_distribute){
                // Making data
                case 1: //uniform
                {
                        for (int i = 0; i < M; i++) {
                                x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
                                y[i] = RESCALE(M_PI*randm11(), nf2, 1);
                                c[i].real() = randm11();
                                c[i].imag() = randm11();
                        }
                }
                break;
                case 2: // concentrate on a small region
                {
                        for (int i = 0; i < M; i++) {
                                x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);// x in [-pi,pi)
                                y[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf2, 1);
                                c[i].real() = randm11();
                                c[i].imag() = randm11();
                        }
                        ofstream xyfile ("nupts_location_nonuniform.txt");

                }
                break;
        }

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"<<endl;
#endif
	if(opts.method == 3)
	{
		opts.bin_size_x=4;
		opts.bin_size_y=4;
	}

	if(opts.method == 4)
	{
		opts.bin_size_x=32;
		opts.bin_size_y=32;
	}

	timer.restart();
	ier = cnufftspread2d_gpu(nf1, nf2, fw, M, x, y, c, opts);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (\t%.3g NU pts/s)\n",
		opts.method,M,nf1*nf2,t,M/t);
#ifdef RESULT
	switch(method)
	{
		case 3:
			opts.bin_size_x=4;
			opts.bin_size_y=4;
		case 4:
			opts.bin_size_x=32;
			opts.bin_size_y=32;
	}
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % opts.bin_size_y == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % opts.bin_size_x == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
