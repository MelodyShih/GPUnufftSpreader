#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/spread.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2;
	FLT sigma = 2.0;
	int N1, N2, M;
	if (argc<4) {
		fprintf(stderr,"Usage: spread2d [method [nupts_distr [N1 N2 [M [tol]]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: input driven without sorting\n");
		fprintf(stderr,"method 2: input driven with sorting\n");
		fprintf(stderr,"method 4: hybrid\n");
		fprintf(stderr,"method 5: subprob\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	M = N1*N2;// let density always be 1
	if(argc>5){
		sscanf(argv[5],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		if(M == 0) M=N1*N2;
	}

	FLT tol=1e-6;
	if(argc>6){
		sscanf(argv[6],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int ier;

	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_opts opts;
	cufinufft_plan dplan;
	FLT upsampfac=2.0;

	ier = setup_cuspreader(opts,tol,upsampfac);
        if(ier != 0 ){
                cout<<"error: setup_cuspreader"<<endl;
                return 0;
        }
	opts.spread_direction=1;
	opts.method=method;
	cout<<scientific<<setprecision(3);


	FLT *x, *y;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(CPX));
	hipHostMalloc(&y, M*sizeof(CPX));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*sizeof(CPX));

	opts.pirange=0;
        switch(nupts_distribute){
                // Making data
                case 1: //uniform
                {
                        for (int i = 0; i < M; i++) {
                                x[i] = RESCALE(M_PI*randm11(), nf1, 1);// x in [-pi,pi)
                                y[i] = RESCALE(M_PI*randm11(), nf2, 1);
                                c[i].real() = randm11();
                                c[i].imag() = randm11();
                        }
                }
                break;
                case 2: // concentrate on a small region
                {
                        for (int i = 0; i < M; i++) {
                                x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);// x in [-pi,pi)
                                y[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf2, 1);
                                c[i].real() = randm11();
                                c[i].imag() = randm11();
                        }
                }
                break;
        }

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() <<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"<<endl;
#endif
	if(opts.method == 2)
	{
		opts.bin_size_x=16;
		opts.bin_size_y=16;
	}

	if(opts.method == 4 || opts.method==5)
	{
		opts.bin_size_x=32;
		opts.bin_size_y=32;
	}

	timer.restart();
	ier = cufinufft_spread2d(N1, N2, nf1, nf2, fw, M, x, y, c, opts, &dplan);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (\t%.3g NU pts/s)\n",
		opts.method,M,nf1*nf2,t,M/t);
#ifdef RESULT
	switch(method)
	{
		case 4:
			opts.bin_size_x=32;
			opts.bin_size_y=32;
		case 5:
			opts.bin_size_x=32;
			opts.bin_size_y=32;
		default:
			opts.bin_size_x=nf1;
			opts.bin_size_y=nf2;		
	}
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % opts.bin_size_y == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % opts.bin_size_x == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
