#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include "../src/cufinufft.h"
#include "../src/spreadinterp.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	int nf1, nf2, nf3;
	FLT sigma = 2.0;
	int N1, N2, N3, M;
	if (argc<6) {
		fprintf(stderr,
			"Usage: spread3d [method [nupts_distr [N1 N2 [M [tol [Horner]]]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: input driven without sorting\n");
		fprintf(stderr,"method 5: subprob\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	int nupts_distribute;
	sscanf(argv[2],"%d",&nupts_distribute);
	sscanf(argv[3],"%lf",&w); nf1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[4],"%lf",&w); nf2 = (int)w;  // so can read 1e6 right!
	sscanf(argv[5],"%lf",&w); nf3 = (int)w;  // so can read 1e6 right!

	N1 = (int) nf1/sigma;
	N2 = (int) nf2/sigma;
	N3 = (int) nf3/sigma;
	M = N1*N2*N3;// let density always be 1
	if(argc>6){
		sscanf(argv[6],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
		if(M == 0) M=N1*N2;
	}

	FLT tol=1e-6;
	if(argc>7){
		sscanf(argv[7],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}

	int Horner=0;
	if(argc>8){
		sscanf(argv[8],"%d",&Horner);
	}

	int ier;

	int ns=std::ceil(-log10(tol/10.0));
	cufinufft_opts opts;
	cufinufft_plan dplan;
	FLT upsampfac=2.0;

	ier = cufinufft_default_opts(opts,tol,upsampfac);
	if(ier != 0 ){
		cout<<"error: cufinufft_default_opts"<<endl;
		return 0;
	}
	opts.method=method;
	cout<<scientific<<setprecision(3);


	FLT *x, *y, *z;
	CPX *c, *fw;
	hipHostMalloc(&x, M*sizeof(FLT));
	hipHostMalloc(&y, M*sizeof(FLT));
	hipHostMalloc(&z, M*sizeof(FLT));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fw,nf1*nf2*nf3*sizeof(CPX));

	opts.rescaled=1;
	opts.Horner=Horner;
	switch(nupts_distribute){
		// Making data
		case 1: //uniform
			{
				x[0] = 5;
				y[0] = 5;
				z[0] = 5;
				for (int i = 1; i < M; i++) {
					x[i] = x[0];//RESCALE(M_PI*randm11(), nf1, 1);
					y[i] = y[0];//RESCALE(M_PI*randm11(), nf2, 1);
					z[i] = z[0];//RESCALE(M_PI*randm11(), nf3, 1);
					cout<<z[i]<<endl;
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
				x[1] = 2;
				y[1] = 2;
				z[1] = 2;
			}
			break;
		case 2: // concentrate on a small region
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*rand01()/(nf1*2/32), nf1, 1);
					y[i] = RESCALE(M_PI*rand01()/(nf2*2/32), nf2, 1);
					z[i] = RESCALE(M_PI*rand01()/(nf3*2/32), nf3, 1);
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
		case 3:
			{
				for (int i = 0; i < M; i++) {
					x[i] = RESCALE(M_PI*randm11(), nf1, 1);
					y[i] = RESCALE(M_PI*randm11(), nf2, 1);
					z[i] = RESCALE(M_PI*randm11(), nf3, 1);
					c[i].real() = randm11();
					c[i].imag() = randm11();
				}
			}
			break;
	}

	CNTime timer;
	/*warm up gpu*/
	char *a;
	timer.restart();
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	cout<<"[time  ]"<< " (warm up) First cudamalloc call " << timer.elapsedsec() 
		<<" s"<<endl<<endl;
#endif

#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"
		<<endl;
#endif

	if(opts.method==5)
	{
		opts.bin_size_x=2;
		opts.bin_size_y=2;
		opts.bin_size_z=2;
		opts.o_bin_size_x=8;
		opts.o_bin_size_y=8;
		opts.o_bin_size_z=8;
		opts.maxsubprobsize=2;
	}

	timer.restart();
	ier = cufinufft_spread3d(N1, N2, N3, nf1, nf2, nf3, fw, M, x, y, z, c, 
		opts, &dplan);
	if(ier != 0 ){
		cout<<"error: cnufftspread3d"<<endl;
		return 0;
	}
	FLT t=timer.elapsedsec();
	printf("[Method %d] %ld NU pts to #%d U pts in %.3g s (\t%.3g NU pts/s)\n",
			opts.method,M,nf1*nf2,t,M/t);
#ifdef RESULT
	switch(method)
	{
		case 4:
			opts.bin_size_x=32;
			opts.bin_size_y=32;
		case 5:
			opts.bin_size_x=32;
			opts.bin_size_y=32;
		default:
			opts.bin_size_x=nf1;
			opts.bin_size_y=nf2;		
	}
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		if( j % opts.bin_size_y == 0)
			printf("\n");
		for (int i=0; i<nf1; i++){
			if( i % opts.bin_size_x == 0 && i!=0)
				printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
	cout<<endl;
#endif

	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fw);
	return 0;
}
