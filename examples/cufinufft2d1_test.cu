#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>

#include "../src/spreadinterp.h"
#include "../src/cufinufft.h"
#include "../finufft/utils.h"

using namespace std;

int main(int argc, char* argv[])
{
	FLT sigma = 2.0;
	int N1, N2, M, N;
	if (argc<4) {
		fprintf(stderr,"Usage: cufinufft2d1_test [method [N1 N2 [M [tol]]]]\n");
		fprintf(stderr,"Details --\n");
		fprintf(stderr,"method 1: input driven without sorting\n");
		fprintf(stderr,"method 2: input driven with sorting\n");
		fprintf(stderr,"method 4: hybrid\n");
		fprintf(stderr,"method 5: subprob\n");
		return 1;
	}  
	double w;
	int method;
	sscanf(argv[1],"%d",&method);
	sscanf(argv[2],"%lf",&w); N1 = (int)w;  // so can read 1e6 right!
	sscanf(argv[3],"%lf",&w); N2 = (int)w;  // so can read 1e6 right!
	N = N1*N2;
	M = N1*N2;// let density always be 1
	if(argc>4){
		sscanf(argv[4],"%lf",&w); M  = (int)w;  // so can read 1e6 right!
	}

	FLT tol=1e-6;
	if(argc>5){
		sscanf(argv[5],"%lf",&w); tol  = (FLT)w;  // so can read 1e6 right!
	}
	int iflag=1;


	cout<<scientific<<setprecision(3);
	int ier;


	FLT *x, *y;
	CPX *c, *fk;
	hipHostMalloc(&x, M*sizeof(CPX));
	hipHostMalloc(&y, M*sizeof(CPX));
	hipHostMalloc(&c, M*sizeof(CPX));
	hipHostMalloc(&fk,N1*N2*sizeof(CPX));

	// Making data
	for (int i = 0; i < M; i++) {
		x[i] = M_PI*randm11();// x in [-pi,pi)
		y[i] = M_PI*randm11();
		c[i].real() = randm11();
		c[i].imag() = randm11();
	}
	// This must be here, since in gpu code, x, y gets modified if pirange=1
	int nt1 = (int)(0.37*N1), nt2 = (int)(0.26*N2);  // choose some mode index to check
	CPX Ft = CPX(0,0), J = IMA*(FLT)iflag;
	for (BIGINT j=0; j<M; ++j)
		Ft += c[j] * exp(J*(nt1*x[j]+nt2*y[j]));   // crude direct
	int it = N1/2+nt1 + N1*(N2/2+nt2);   // index in complex F as 1d array

	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*warm up gpu*/
	hipEventRecord(start);
	char *a;
	checkCudaErrors(hipMalloc(&a,1));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tWarm up GPU \t\t %.3g s\n", milliseconds/1000);
#endif

	cufinufft_plan dplan;
	cufinufft_opts opts;
	ier=cufinufft_default_opts(opts,tol,sigma);
	opts.method=method;
	opts.spread_direction=1;

	hipEventRecord(start);
	ier=cufinufft2d_plan(M, x, y, c, N1, N2, fk, iflag, opts, &dplan);
	if (ier!=0){
		printf("err: cufinufft2d_plan\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft plan:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft2d1_exec(opts, &dplan);
	if (ier!=0){
		printf("err: cufinufft2d1_exec\n");
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft exec:\t\t %.3g s\n", milliseconds/1000);

	hipEventRecord(start);
	ier=cufinufft2d_destroy(opts, &dplan);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] cufinufft destroy:\t\t %.3g s\n", milliseconds/1000);
	printf("[gpu   ] one mode: abs err in F[%ld,%ld] is %.3g\n",(int)nt1,(int)nt2,abs(Ft-fk[it]));
	printf("[gpu   ] one mode: rel err in F[%ld,%ld] is %.3g\n",(int)nt1,(int)nt2,abs(Ft-fk[it])/infnorm(N,fk));
#if 0
	cout<<"[result-input]"<<endl;
	for(int j=0; j<nf2; j++){
		//        if( j % opts.bin_size_y == 0)
		//                printf("\n");
		for (int i=0; i<nf1; i++){
			//                if( i % opts.bin_size_x == 0 && i!=0)
			//                        printf(" |");
			printf(" (%2.3g,%2.3g)",fw[i+j*nf1].real(),fw[i+j*nf1].imag() );
		}
		cout<<endl;
	}
#endif	
	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(c);
	hipHostFree(fk);
	return 0;
}
