#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
// idriven coarse grained
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "spread.h"

using namespace std;

int cnufft_allocgpumemory(int nf1, int nf2, int M, int* fw_width, CPX* h_fw, gpuComplex** d_fw, 
		FLT *h_kx, FLT **d_kx, FLT* h_ky, FLT** d_ky, 
		CPX *h_c, gpuComplex **d_c)
{
	checkCudaErrors(hipMalloc(d_kx,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(d_ky,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(d_c,M*sizeof(gpuComplex)));

	size_t pitch;
	checkCudaErrors(hipMallocPitch((void**) d_fw, &pitch,nf1*sizeof(gpuComplex),nf2));
	*fw_width = pitch/sizeof(gpuComplex);

	return 0;
}

int cnufft_copycpumem_to_gpumem(int nf1, int nf2, int M, int fw_width, CPX* h_fw, gpuComplex* d_fw,
		FLT *h_kx, FLT *d_kx, FLT* h_ky, FLT* d_ky,
		CPX *h_c, gpuComplex *d_c)
{
	checkCudaErrors(hipMemcpy(d_kx,h_kx,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ky,h_ky,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c,h_c,M*sizeof(gpuComplex),hipMemcpyHostToDevice));

	return 0;
}

int cnufft_copygpumem_to_cpumem(int nf1, int nf2, int M, int fw_width, CPX* h_fw, gpuComplex* d_fw,
		FLT *h_kx, FLT *d_kx, FLT* h_ky, FLT* d_ky,
		CPX *h_c, gpuComplex *d_c)
{
	checkCudaErrors(hipMemcpy2D(h_fw,nf1*sizeof(gpuComplex),d_fw,fw_width*sizeof(gpuComplex),
				nf1*sizeof(gpuComplex),nf2,hipMemcpyDeviceToHost));

	return 0;
}

void cnufft_free_gpumemory(gpuComplex* d_fw, FLT *d_kx, FLT* d_ky, gpuComplex *d_c)
{
	hipFree(d_fw);
	hipFree(d_kx);
	hipFree(d_ky);
	hipFree(d_c);
}

int cnufftspread2d_gpu(int nf1, int nf2, CPX* h_fw, int M, FLT *h_kx,
		FLT *h_ky, CPX *h_c, spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	int fw_width;
	FLT *d_kx,*d_ky;
	gpuComplex *d_c,*d_fw;

	if(opts.pirange){
		for(int i=0; i<M; i++){
			h_kx[i]=RESCALE(h_kx[i], nf1, opts.pirange);
			h_ky[i]=RESCALE(h_ky[i], nf2, opts.pirange);
		}
	}
	hipEventRecord(start);
	ier = cnufft_allocgpumemory(nf1, nf2, M, &fw_width, h_fw, &d_fw, h_kx, &d_kx, 
			h_ky, &d_ky, h_c, &d_c);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Allocating GPU memory " << milliseconds <<" ms"<<endl;
#endif
	hipEventRecord(start);
	ier = cnufft_copycpumem_to_gpumem(nf1, nf2, M, fw_width, h_fw, d_fw, h_kx, d_kx,
			h_ky, d_ky, h_c, d_c);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Copying memory from host to device " << milliseconds <<" s"<<endl;
#endif

	switch(opts.method)
	{
		case 1:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_idriven(nf1, nf2, fw_width, d_fw, M, d_kx, 
						d_ky, d_c, opts);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_idriven"<<endl;
					return 0;
				}
			}
			break;
		case 2:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_idriven_sorted(nf1, nf2, fw_width, d_fw, M, 
						d_kx, d_ky, d_c, opts);
			}
			break;
		case 3:
			{
				hipEventRecord(start);
				if(nf1 % opts.bin_size_x != 0 || nf2 % opts.bin_size_y !=0){
					cout << "error: mod(nf1,block_size_x) and mod(nf2,block_size_y) should be 0" << endl;
					return 0;
				}
				ier = cnufftspread2d_gpu_odriven(nf1, nf2, fw_width, d_fw, M, d_kx, 
						d_ky, d_c, opts);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_odriven"<<endl;
					return 0;
				}
			}
			break;
		case 4:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_hybrid(nf1, nf2, fw_width, d_fw, M, d_kx, d_ky, d_c, opts);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_hybrid"<<endl;
					return 0;
				}
			}
			break;
		case 5:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_subprob(nf1, nf2, fw_width, d_fw, M, d_kx, d_ky, d_c, opts);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_hybrid"<<endl;
					return 0;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1,2,3 or 4"<<endl;
			return 0;
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Spread " << milliseconds <<" ms"<<endl;
#endif
	hipEventRecord(start);
	ier = cnufft_copygpumem_to_cpumem(nf1, nf2, M, fw_width, h_fw, d_fw, h_kx, d_kx,
			h_ky, d_ky, h_c, d_c);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Copying memory from device to host " << milliseconds <<" ms"<<endl;
#endif
	cnufft_free_gpumemory(d_fw, d_kx, d_ky, d_c);

	return ier;
}

int cnufftspread2d_gpu_simple(int nf1, int nf2, int fw_width, gpuComplex* d_fw, int M, FLT *d_kx,
		FLT *d_ky, gpuComplex *d_c, spread_opts opts, int binx, int biny)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;

	// assume that bin_size_x > ns/2;
	hipEventRecord(start);
	threadsPerBlock.x = opts.nthread_x;
	threadsPerBlock.y = opts.nthread_y;
	blocks.x = 1;
	blocks.y = 1;
	size_t sharedmemorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(gpuComplex);
	if(sharedmemorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}
	// blockSize must be a multiple of bin_size_x
	Spread_2d_Simple<<<blocks, threadsPerBlock, sharedmemorysize>>>(d_kx, d_ky, d_c, 
			d_fw, M, ns, nf1, nf2, 
			es_c, es_beta, fw_width, 
			M, bin_size_x, bin_size_y, 
			binx, biny);
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Simple \t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_idriven(int nf1, int nf2, int fw_width, gpuComplex* d_fw, int M, FLT *d_kx,
		FLT *d_ky, gpuComplex *d_c, spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	hipEventRecord(start);
	if(opts.Horner){
		Spread_2d_Idriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, d_fw, M, ns,
				nf1, nf2, es_c, es_beta, fw_width);
	}else{
		Spread_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, d_fw, M, ns,
				nf1, nf2, es_c, es_beta, fw_width);
	}

#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Idriven \t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_idriven_sorted(int nf1, int nf2, int fw_width, gpuComplex* d_fw, 
		int M, FLT *d_kx, FLT *d_ky, gpuComplex *d_c, 
		spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;

	FLT *d_kxsorted,*d_kysorted;
	gpuComplex *d_csorted;

	// following variables are used when bin_sort=1
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int numbins[2];
	int *d_binsize;
	int *d_binstartpts;

	// following variables are used when bin_sort=0
	int *d_sortedidx;
	int *d_index_out, *d_index_in;
	
	// following variables are used both in bin_sort=0 and 1 case
	int *d_sortidx;
	void*d_temp_storage=NULL;

	hipEventRecord(start);
	checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));
	checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
	if(opts.bin_sort){
		numbins[0] = ceil((FLT) nf1/bin_size_x);
		numbins[1] = ceil((FLT) nf2/bin_size_y);
		checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
		checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1])*sizeof(int)));
	}else{
		checkCudaErrors(hipMalloc(&d_sortedidx,M*sizeof(int)));
		checkCudaErrors(hipMalloc(&d_index_in,M*sizeof(int)));
		checkCudaErrors(hipMalloc(&d_index_out,M*sizeof(int)));
	}

#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocating GPU memory for sorted array \t%.3g ms\n", milliseconds);
#endif
	if(opts.bin_sort){
		hipEventRecord(start);
		checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
		CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
				numbins[0],numbins[1],d_binsize,
				d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
		hipEventRecord(start);
		int n=numbins[0]*numbins[1];
		size_t temp_storage_bytes = 0;
		CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
		checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
		CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts+1, n));
#ifdef SPREADTIME
        	hipEventRecord(stop);
        	hipEventSynchronize(stop);
        	hipEventElapsedTime(&milliseconds, start, stop);
        	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
		hipEventRecord(start);
		PtsRearrage_noghost_2d<<<(M+1024-1)/1024,1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
				numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted,
				d_ky, d_kysorted, d_c, d_csorted);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel PtsRearrange_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
	}else{
		hipEventRecord(start);
		threadsPerBlock.x = 1024;
		threadsPerBlock.y = 1;
		blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
		blocks.y = 1;
		CreateSortIdx<<<blocks, threadsPerBlock>>>(M, nf1, nf2, d_kx, d_ky, d_sortidx);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCreateSortIdx \t\t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
		FLT *h_kx, *h_ky;
		CPX *h_c;
		h_kx = (FLT*) malloc(M*sizeof(FLT)); 
		h_ky = (FLT*) malloc(M*sizeof(FLT));
		h_c = (CPX*) malloc(M*sizeof(CPX));
		checkCudaErrors(hipMemcpy(h_kx, d_kx, M*sizeof(FLT), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_ky, d_ky, M*sizeof(FLT), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_c, d_c, M*sizeof(CPX), hipMemcpyDeviceToHost));
		int* h_sortidx = (int*) malloc(M*sizeof(int));
		checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),hipMemcpyDeviceToHost));
		for(int i=0; i<M; i++){
			printf("sortidx = %d, (x,y) = (%.3g, %.3g), c=(%f, %f)\n", h_sortidx[i], h_kx[i], 
					h_ky[i], h_c[i].real(), 
					h_c[i].imag());
		}
		free(h_sortidx);
#endif 
		hipEventRecord(start);
		size_t  temp_storage_bytes  = 0;

		threadsPerBlock.x = 1024;
		threadsPerBlock.y = 1;
		blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
		blocks.y = 1;
		CreateIndex<<<blocks, threadsPerBlock>>>(d_index_in, M);
		hipEventRecord(start);
		hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_sortidx, 
				d_sortedidx, d_index_in, d_index_out, M);
		checkCudaErrors(hipMalloc(&d_temp_storage,temp_storage_bytes));
		hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_sortidx, 
				d_sortedidx, d_index_in, d_index_out, M);

#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCUB::SortPairs \t\t\t\t%.3g ms\n", milliseconds);
#endif
		hipEventRecord(start);
		threadsPerBlock.x = 1024;
		threadsPerBlock.y = 1;
		blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
		blocks.y = 1;
		Gather<<<blocks, threadsPerBlock>>>(M, d_index_out, d_kx, d_ky, d_c, d_kxsorted, d_kysorted, d_csorted);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel (Gather) PtsRearrage \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
		checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_kx,d_kxsorted,M*sizeof(FLT),hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_ky,d_kysorted,M*sizeof(FLT),hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_c,d_csorted,M*sizeof(gpuComplex),hipMemcpyDeviceToHost));
		for(int i=0; i<M; i++){
			printf("sortidx = %d, (x,y) = (%.3g, %.3g), c=(%f, %f)\n", h_sortidx[i], h_kx[i], h_ky[i], h_c[i].real(), h_c[i].imag());
		}
#endif 
	}
	hipEventRecord(start);
	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	Spread_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kxsorted, d_kysorted, d_csorted, d_fw, M, ns,
			nf1, nf2, es_c, es_beta, fw_width);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Idriven \t\t%.3g ms\n", milliseconds);
#endif
	// Free memory
	hipEventRecord(start);
	hipFree(d_kxsorted);
	hipFree(d_kysorted);
	hipFree(d_csorted);
	hipFree(d_sortidx);
	if(opts.bin_sort){
		hipFree(d_binsize);
		hipFree(d_binstartpts);
		hipFree(d_temp_storage);
	}else{
		hipFree(d_sortedidx);
		hipFree(d_index_in);
		hipFree(d_index_out);
		hipFree(d_temp_storage);
	}
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree part GPU-memory \t\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_hybrid(int nf1, int nf2, int fw_width, gpuComplex* d_fw, 
		int M, FLT *d_kx, FLT *d_ky, gpuComplex *d_c, 
		spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;

	// Parameter setting
	int numbins[2];

	int *d_binsize;
	int *d_binstartpts;
	int *d_sortidx;

	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
	// assume that bin_size_x > ns/2;
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif
	FLT *d_kxsorted,*d_kysorted;
	gpuComplex *d_csorted;


	hipEventRecord(start);
	checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));

	checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocating GPU memory for sorted array \t%.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
			numbins[0],numbins[1],d_binsize,
			d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize;// For debug
	h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binsize);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
#if 0
	int n=numbins[0]*numbins[1];
	int scanblocksize=1024;
	int numscanblocks=ceil((double)n/scanblocksize);
	int* d_scanblocksum, *d_scanblockstartpts;
#ifdef DEBUG
	printf("[debug ] n=%d, numscanblocks=%d\n",n,numscanblocks);
#endif 
	checkCudaErrors(hipMalloc(&d_scanblocksum,numscanblocks*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_scanblockstartpts,(numscanblocks+1)*sizeof(int)));

	for(int i=0;i<numscanblocks;i++){
		int nelemtoscan=(n-scanblocksize*i)>scanblocksize ? scanblocksize : n-scanblocksize*i;
		prescan<<<1, scanblocksize/2>>>(nelemtoscan,d_binsize+i*scanblocksize,
				d_binstartpts+i*scanblocksize,d_scanblocksum+i);
	}
#ifdef DEBUG
	int* h_scanblocksum;
	h_scanblocksum     =(int*) malloc(numscanblocks*sizeof(int));
	checkCudaErrors(hipMemcpy(h_scanblocksum,d_scanblocksum,numscanblocks*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int i=0;i<numscanblocks;i++){
		cout<<"[debug ] scanblocksum["<<i<<"]="<<h_scanblocksum[i]<<endl;
	}
#endif
	int next = pow(2, ceil(log(numscanblocks+1)/log(2)));
	if(next > 2048){
		cout<<"error: number of elements to sort exceed the prescan capability"<<endl;
		return 1;
	}
	prescan<<<1, next/2>>>(numscanblocks,d_scanblocksum,d_scanblockstartpts,d_scanblockstartpts+numscanblocks);
#ifdef DEBUG
	int* h_scanblockstartpts = (int*) malloc((numscanblocks+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_scanblockstartpts,d_scanblockstartpts,(numscanblocks+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int i=0;i<numscanblocks+1;i++){
		cout<<"[debug ] scanblockstartpts["<<i<<"]="<<h_scanblockstartpts[i]<<endl;
	}
#endif
	uniformUpdate<<<numscanblocks,scanblocksize>>>(n,d_binstartpts,d_scanblockstartpts);
#endif
	int n=numbins[0]*numbins[1];
	void *d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts+1, n));
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes)); // Allocate temporary storage for inclusive prefix scan
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts+1, n));
	checkCudaErrors(hipMemset(d_binstartpts,0,sizeof(int)));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc((numbins[0]*numbins[1]+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_binstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	cout<<"[debug ] Total number of nonuniform pts (include those in ghost bins) = "
		<< setw(4)<<h_binstartpts[numbins[0]*numbins[1]]<<endl;
	free(h_binstartpts);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	PtsRearrage_noghost_2d<<<(M+1024-1)/1024,1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
			numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted,
			d_ky, d_kysorted, d_c, d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel PtsRearrange_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	FLT *h_kxsorted, *h_kysorted;
	CPX *h_csorted;
	h_kxsorted = (FLT*)malloc(M*sizeof(FLT));
	h_kysorted = (FLT*)malloc(M*sizeof(FLT));
	h_csorted  = (CPX*)malloc(M*sizeof(CPX));
	checkCudaErrors(hipMemcpy(h_kxsorted,d_kxsorted,M*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_kysorted,d_kysorted,M*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_csorted,d_csorted,M*sizeof(CPX),
				hipMemcpyDeviceToHost));
	for (int i=0; i<10; i++){
		cout <<"[debug ] (x,y) = ("<<setw(10)<<h_kxsorted[i]<<","
			<<setw(10)<<h_kysorted[i]<<"), bin# =  "
			<<(floor(h_kxsorted[i]/bin_size_x))+numbins[0]*(floor(h_kysorted[i]/bin_size_y))<<endl;
	}
	free(h_kysorted);
	free(h_kxsorted);
	free(h_csorted);
#endif

	hipEventRecord(start);
	threadsPerBlock.x = 16;
	threadsPerBlock.y = 16;
	blocks.x = numbins[0];
	blocks.y = numbins[1];
	size_t sharedmemorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(gpuComplex);
	if(sharedmemorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}
	// blockSize must be a multiple of bin_size_x
	Spread_2d_Hybrid<<<blocks, threadsPerBlock, sharedmemorysize>>>(d_kxsorted, d_kysorted, d_csorted, 
			d_fw, M, ns, nf1, nf2, 
			es_c, es_beta, fw_width, 
			d_binstartpts, d_binsize, 
			bin_size_x, bin_size_y);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Hybrid \t\t%.3g ms\n", milliseconds);
#endif
	// Free memory
	hipFree(d_temp_storage);
	hipEventRecord(start);
	hipFree(d_binsize);
	hipFree(d_binstartpts);
	hipFree(d_sortidx);
	hipFree(d_kxsorted);
	hipFree(d_kysorted);
	hipFree(d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree part GPU-memory \t\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_odriven(int nf1, int nf2, int fw_width, gpuComplex* d_fw, int M, 
		FLT *d_kx, FLT *d_ky, gpuComplex *d_c, spread_opts opts)
{
	// Timing 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;

	// GPU memory
	FLT *d_kxsorted,*d_kysorted;
	gpuComplex *d_csorted;
	int *d_binsize;
	int *d_binstartpts;
	int *d_sortidx;

	// Parameter setting
	int numbins[2];
	int totalnupts;
	int nbin_block_x, nbin_block_y;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;

	numbins[0] = ceil(nf1/bin_size_x)+2;
	numbins[1] = ceil(nf2/bin_size_y)+2;
	// assume that bin_size_x > ns/2;
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins (including ghost bins) = ["
		<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

	checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));


	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	hipEventRecord(start);
	hipEventRecord(start);
	CalcBinSize_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
			numbins[0],numbins[1],d_binsize,
			d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize; // For debug
	h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Before fill in the ghost bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif
	hipEventRecord(start);
	threadsPerBlock.x = 32;
	threadsPerBlock.y = 32;
	if(threadsPerBlock.x*threadsPerBlock.y < 1024){
		cout<<"error: number of threads in a block exceeds max num 1024("
			<<threadsPerBlock.x*threadsPerBlock.y<<")"<<endl;
		return 1;
	}
	blocks.x = (numbins[0]+threadsPerBlock.x-1)/threadsPerBlock.x;
	blocks.y = (numbins[1]+threadsPerBlock.y-1)/threadsPerBlock.y;
	FillGhostBin_2d<<<blocks,threadsPerBlock>>>(numbins[0],numbins[1],d_binsize);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel FillGhostBin_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] After fill in the ghost bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binsize);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	int n=numbins[0]*numbins[1];
	int scanblocksize=1024;
	int numscanblocks=ceil((double)n/scanblocksize);
	int* d_scanblocksum, *d_scanblockstartpts;
#ifdef DEBUG
	printf("[debug ] n=%d, numscanblocks=%d\n",n,numscanblocks);
#endif 
	checkCudaErrors(hipMalloc(&d_scanblocksum,numscanblocks*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_scanblockstartpts,(numscanblocks+1)*sizeof(int)));

	for(int i=0;i<numscanblocks;i++){
		int nelemtoscan=(n-scanblocksize*i)>scanblocksize ? scanblocksize : n-scanblocksize*i;
		prescan<<<1, scanblocksize/2>>>(nelemtoscan,d_binsize+i*scanblocksize,
				d_binstartpts+i*scanblocksize,d_scanblocksum+i);
	}
#ifdef DEBUG
	int* h_scanblocksum;
	h_scanblocksum     =(int*) malloc(numscanblocks*sizeof(int));
	checkCudaErrors(hipMemcpy(h_scanblocksum,d_scanblocksum,numscanblocks*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int i=0;i<numscanblocks;i++){
		cout<<"[debug ] scanblocksum["<<i<<"]="<<h_scanblocksum[i]<<endl;
	}
#endif
	int next = pow(2, ceil(log(numscanblocks+1)/log(2)));
	if(next > 2048){
		cout<<"error: number of elements to sort exceed the prescan capability"<<endl;
		return 1;
	}
	prescan<<<1, next/2>>>(numscanblocks,d_scanblocksum,d_scanblockstartpts,d_scanblockstartpts+numscanblocks);
#ifdef DEBUG
	int* h_scanblockstartpts = (int*) malloc((numscanblocks+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_scanblockstartpts,d_scanblockstartpts,(numscanblocks+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int i=0;i<numscanblocks+1;i++){
		cout<<"[debug ] scanblockstartpts["<<i<<"]="<<h_scanblockstartpts[i]<<endl;
	}
#endif
	uniformUpdate<<<numscanblocks,scanblocksize>>>(n,d_binstartpts,d_scanblockstartpts);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc((numbins[0]*numbins[1]+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_binstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	cout<<"[debug ] Total number of nonuniform pts (include those in ghost bins) = "
		<< setw(4)<<h_binstartpts[numbins[0]*numbins[1]]<<endl;
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
	free(h_binstartpts);
#endif

	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(&totalnupts,d_binstartpts+numbins[0]*numbins[1],sizeof(int),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMalloc(&d_kxsorted,totalnupts*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_kysorted,totalnupts*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_csorted,totalnupts*sizeof(gpuComplex)));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocating GPU memory for sorted array \t%.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	PtsRearrage_2d<<<(M+1024-1)/1024,1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
			numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted,
			d_ky, d_kysorted, d_c, d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel PtsRearrange_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	FLT *h_kxsorted, *h_kysorted;
	CPX *h_csorted;
	h_kxsorted = (FLT*)malloc(totalnupts*sizeof(FLT));
	h_kysorted = (FLT*)malloc(totalnupts*sizeof(FLT));
	h_csorted  = (CPX*)malloc(totalnupts*sizeof(CPX));
	checkCudaErrors(hipMemcpy(h_kxsorted,d_kxsorted,totalnupts*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_kysorted,d_kysorted,totalnupts*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_csorted,d_csorted,totalnupts*sizeof(CPX),
				hipMemcpyDeviceToHost));
	for (int i=0; i<totalnupts; i++){
		//printf("[debug ] (x,y)=(%f, %f), bin#=%d\n", h_kxsorted[i], h_kysorted[i],
		//                                             (floor(h_kxsorted[i]/bin_size_x)+1)+numbins[0]*(floor(h_kysorted[i]/bin_size_y)+1));
		cout <<"[debug ] (x,y) = ("<<setw(10)<<h_kxsorted[i]<<","
			<<setw(10)<<h_kysorted[i]<<"), bin# =  "
			<<(floor(h_kxsorted[i]/bin_size_x)+1)+numbins[0]*(floor(h_kysorted[i]/bin_size_y)+1)<<endl;
	}
	free(h_kysorted);
	free(h_kxsorted);
	free(h_csorted);
#endif

	hipEventRecord(start);
	threadsPerBlock.x = 8;
	threadsPerBlock.y = 8;
	blocks.x = (nf1 + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = (nf2 + threadsPerBlock.y - 1)/threadsPerBlock.y;
	nbin_block_x = threadsPerBlock.x/bin_size_x<(numbins[0]-2) ? threadsPerBlock.x/bin_size_x : (numbins[0]-2);
	nbin_block_y = threadsPerBlock.y/bin_size_y<(numbins[1]-2) ? threadsPerBlock.y/bin_size_y : (numbins[1]-2);
#ifdef INFO
	cout<<"[info  ]"<<" ["<<nf1<<"x"<<nf2<<"] "<<"output elements is divided into ["
		<<blocks.x<<","<<blocks.y<<"] block"<<", each block has ["<<nbin_block_x<<"x"<<nbin_block_y<<"] bins, "
		<<"["<<threadsPerBlock.x<<"x"<<threadsPerBlock.y<<"] threads"<<endl;
#endif
	// blockSize must be a multiple of bin_size_x
	Spread_2d_Odriven<<<blocks, threadsPerBlock>>>(nbin_block_x, nbin_block_y, numbins[0], numbins[1],
			d_binstartpts, d_kxsorted, d_kysorted, d_csorted,
			d_fw, ns, nf1, nf2, es_c, es_beta, fw_width);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Odriven \t\t%.3g ms\n", milliseconds);
#endif
	// Free memory
	hipEventRecord(start);
	hipFree(d_binsize);
	hipFree(d_binstartpts);
	hipFree(d_sortidx);
	hipFree(d_kxsorted);
	hipFree(d_kysorted);
	hipFree(d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree part GPU-memory \t\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_subprob(int nf1, int nf2, int fw_width, gpuComplex* d_fw, 
		int M, FLT *d_kx, FLT *d_ky, gpuComplex *d_c, 
		spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int maxsubprobsize=opts.maxsubprobsize;

	// Parameter setting
	int numbins[2];

	int *d_binsize, *d_binstartpts;
	int *d_sortidx;
	int *d_numsubprob, *d_subprobstartpts;

	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
	// assume that bin_size_x > ns/2;
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif
	FLT *d_kxsorted,*d_kysorted;
	gpuComplex *d_csorted;


	hipEventRecord(start);
	checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));
	checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));

	checkCudaErrors(hipMalloc(&d_numsubprob,  numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binsize,     numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binstartpts, numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_subprobstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocating GPU memory for sorted array \t%.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
			numbins[0],numbins[1],d_binsize,
			d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize;// For debug
	h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binsize);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	int n=numbins[0]*numbins[1];
	void *d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes)); // Allocate temporary storage for inclusive prefix scan
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc((numbins[0]*numbins[1]+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_binstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	cout<<"[debug ] Total number of nonuniform pts (include those in ghost bins) = "
		<< setw(4)<<h_binstartpts[numbins[0]*numbins[1]]<<endl;
	free(h_binstartpts);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	PtsRearrage_noghost_2d<<<(M+1024-1)/1024,1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
			numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted,
			d_ky, d_kysorted, d_c, d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel PtsRearrange_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	FLT *h_kxsorted, *h_kysorted;
	CPX *h_csorted;
	h_kxsorted = (FLT*)malloc(M*sizeof(FLT));
	h_kysorted = (FLT*)malloc(M*sizeof(FLT));
	h_csorted  = (CPX*)malloc(M*sizeof(CPX));
	checkCudaErrors(hipMemcpy(h_kxsorted,d_kxsorted,M*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_kysorted,d_kysorted,M*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_csorted,d_csorted,M*sizeof(CPX),
				hipMemcpyDeviceToHost));
	for (int i=0; i<10; i++){
		cout <<"[debug ] (x,y) = ("<<setw(10)<<h_kxsorted[i]<<","
			<<setw(10)<<h_kysorted[i]<<"), bin# =  "
			<<(floor(h_kxsorted[i]/bin_size_x))+numbins[0]*(floor(h_kysorted[i]/bin_size_y))<<endl;
	}
	free(h_kysorted);
	free(h_kxsorted);
	free(h_csorted);
#endif

	/* --------------------------------------------- */
	//        Determining Subproblem properties        //
	/* --------------------------------------------- */

	hipEventRecord(start);
	CalcSubProb_2d<<<(M+1024-1)/1024, 1024>>>(d_binsize, d_numsubprob,maxsubprobsize,numbins[0]*numbins[1]);
#ifdef DEBUG
	int* h_numsubprob;
	h_numsubprob = (int*) malloc(n*sizeof(int));
	checkCudaErrors(hipMemcpy(h_numsubprob,d_numsubprob,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_numsubprob[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_numsubprob);
#endif
	// Scanning the same length array, so we don't need calculate temp_storage_bytes here
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_numsubprob, d_subprobstartpts+1, n));
	checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));

#ifdef DEBUG
	printf("[debug ] Subproblem start points\n");
	int* h_subprobstartpts;
	h_subprobstartpts = (int*) malloc((n+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprobstartpts,d_subprobstartpts,(n+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_subprobstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
	free(h_subprobstartpts);
#endif

	int totalnumsubprob;
	checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],sizeof(int),
				hipMemcpyDeviceToHost));
	int* d_subprob_to_bin;
	checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
	MapBintoSubProb_2d<<<(numbins[0]*numbins[1]+1024-1)/1024, 1024>>>(d_subprob_to_bin, 
									  d_subprobstartpts,
									  d_numsubprob,
									  numbins[0]*numbins[1]);
#ifdef DEBUG
	printf("[debug ] Map Subproblem to Bins\n");
	int* h_subprob_to_bin;
	h_subprob_to_bin = (int*) malloc((totalnumsubprob)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprob_to_bin,d_subprob_to_bin,(totalnumsubprob)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<totalnumsubprob; j++){
		cout<<"[debug ] ";
		cout <<"nsub["<<j<<"] = "<<setw(2)<<h_subprob_to_bin[j];
		cout<<endl;
	}
	free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	size_t sharedmemorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(gpuComplex);
	if(sharedmemorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}
	// blockSize must be a multiple of bin_size_x
	Spread_2d_Subprob<<<totalnumsubprob, 256, sharedmemorysize>>>(d_kxsorted, d_kysorted, d_csorted, 
								      d_fw, M, ns, nf1, nf2, 
								      es_c, es_beta, fw_width, 
								      d_binstartpts, d_binsize, 
								      bin_size_x, bin_size_y,
								      d_subprob_to_bin, d_subprobstartpts, 
								      d_numsubprob, maxsubprobsize, 
								      numbins[0], numbins[1]);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Subprob \t\t%.3g ms\n", milliseconds);
#endif
	// Free memory
	hipFree(d_temp_storage);
	hipEventRecord(start);
	hipFree(d_binsize);
	hipFree(d_binstartpts);
	hipFree(d_sortidx);
	hipFree(d_kxsorted);
	hipFree(d_kysorted);
	hipFree(d_csorted);
	hipFree(d_numsubprob);
	hipFree(d_subprobstartpts);
	hipFree(d_subprob_to_bin);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree part GPU-memory \t\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}
