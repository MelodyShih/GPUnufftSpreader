#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
// idriven coarse grained
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "spread.h"

using namespace std;

int cnufft_allocgpumemory(int nf1, int nf2, int M, int* fw_width, CPX* h_fw, gpuComplex** d_fw, 
		FLT *h_kx, FLT **d_kx, FLT* h_ky, FLT** d_ky, 
		CPX *h_c, gpuComplex **d_c)
{
#if 0
	__device__ FLT d_kx[];
	__device__ FLT d_ky[];
	__device__ gpuComplex d_kc[];

	// No extra memory is needed in idriven method;
	switch(opts.method)
	{
		case 2:
			{
				int *d_sortidx;
				checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));
				checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));

				int *d_binsize;
				int *d_binstartpts;
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1])*sizeof(int)));
			}
			break;
		case 4:
			{
				FLT *d_kxsorted,*d_kysorted;
				gpuComplex *d_csorted;
				checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));

				int numbins[2];
				int *d_binsize;
				int *d_binstartpts;
				int *d_sortidx;
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
		case 5:
			{

			}
			break;
	}
#endif
	checkCudaErrors(hipMalloc(d_kx,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(d_ky,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(d_c,M*sizeof(gpuComplex)));

	size_t pitch;
	checkCudaErrors(hipMallocPitch((void**) d_fw, &pitch,nf1*sizeof(gpuComplex),nf2));
	*fw_width = pitch/sizeof(gpuComplex);

	return 0;
}

int cnufft_copycpumem_to_gpumem(int nf1, int nf2, int M, int fw_width, CPX* h_fw, gpuComplex* d_fw,
		FLT *h_kx, FLT *d_kx, FLT* h_ky, FLT* d_ky,
		CPX *h_c, gpuComplex *d_c)
{
	checkCudaErrors(hipMemcpy(d_kx,h_kx,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ky,h_ky,M*sizeof(FLT),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c,h_c,M*sizeof(gpuComplex),hipMemcpyHostToDevice));

	return 0;
}

int cnufft_copygpumem_to_cpumem(int nf1, int nf2, int M, int fw_width, CPX* h_fw, gpuComplex* d_fw,
		FLT *h_kx, FLT *d_kx, FLT* h_ky, FLT* d_ky,
		CPX *h_c, gpuComplex *d_c)
{
	checkCudaErrors(hipMemcpy2D(h_fw,nf1*sizeof(gpuComplex),d_fw,fw_width*sizeof(gpuComplex),
				nf1*sizeof(gpuComplex),nf2,hipMemcpyDeviceToHost));

	return 0;
}

void cnufft_free_gpumemory(gpuComplex* d_fw, FLT *d_kx, FLT* d_ky, gpuComplex *d_c)
{
	hipFree(d_fw);
	hipFree(d_kx);
	hipFree(d_ky);
	hipFree(d_c);
}

int cnufftspread2d_gpu(int nf1, int nf2, CPX* h_fw, int M, FLT *h_kx,
		FLT *h_ky, CPX *h_c, spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	int fw_width;
	FLT *d_kx,*d_ky;
	gpuComplex *d_c,*d_fw;

	if(opts.pirange){
		for(int i=0; i<M; i++){
			h_kx[i]=RESCALE(h_kx[i], nf1, opts.pirange);
			h_ky[i]=RESCALE(h_ky[i], nf2, opts.pirange);
		}
	}
	hipEventRecord(start);
	ier = cnufft_allocgpumemory(nf1, nf2, M, &fw_width, h_fw, &d_fw, h_kx, &d_kx, 
			h_ky, &d_ky, h_c, &d_c);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Allocating GPU memory " << milliseconds <<" ms"<<endl;
#endif
	hipEventRecord(start);
	ier = cnufft_copycpumem_to_gpumem(nf1, nf2, M, fw_width, h_fw, d_fw, h_kx, d_kx,
			h_ky, d_ky, h_c, d_c);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Copying memory from host to device " << milliseconds <<" s"<<endl;
#endif

	switch(opts.method)
	{
		case 1:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_idriven(nf1, nf2, fw_width, d_fw, M, d_kx, 
						d_ky, d_c, opts);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_idriven"<<endl;
					return 0;
				}
			}
			break;
		case 2:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_idriven_sorted(nf1, nf2, fw_width, d_fw, M, 
						d_kx, d_ky, d_c, opts);
			}
			break;
		case 4:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_hybrid(nf1, nf2, fw_width, d_fw, M, d_kx, d_ky, d_c, opts);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_hybrid"<<endl;
					return 0;
				}
			}
			break;
		case 5:
			{
				hipEventRecord(start);
				ier = cnufftspread2d_gpu_subprob(nf1, nf2, fw_width, d_fw, M, d_kx, d_ky, d_c, opts);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_hybrid"<<endl;
					return 0;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1,2,4 or 5"<<endl;
			return 0;
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Spread " << milliseconds <<" ms"<<endl;
#endif
	hipEventRecord(start);
	ier = cnufft_copygpumem_to_cpumem(nf1, nf2, M, fw_width, h_fw, d_fw, h_kx, d_kx,
			h_ky, d_ky, h_c, d_c);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Copying memory from device to host " << milliseconds <<" ms"<<endl;
#endif
	cnufft_free_gpumemory(d_fw, d_kx, d_ky, d_c);

	return ier;
}

int cnufftspread2d_gpu_simple(int nf1, int nf2, int fw_width, gpuComplex* d_fw, int M, FLT *d_kx,
		FLT *d_ky, gpuComplex *d_c, spread_opts opts, int binx, int biny)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;

	// assume that bin_size_x > ns/2;
	hipEventRecord(start);
	threadsPerBlock.x = opts.nthread_x;
	threadsPerBlock.y = opts.nthread_y;
	blocks.x = 1;
	blocks.y = 1;
	size_t sharedmemorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(gpuComplex);
	if(sharedmemorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}
	// blockSize must be a multiple of bin_size_x
	Spread_2d_Simple<<<blocks, threadsPerBlock, sharedmemorysize>>>(d_kx, d_ky, d_c, 
			d_fw, M, ns, nf1, nf2, 
			es_c, es_beta, fw_width, 
			M, bin_size_x, bin_size_y, 
			binx, biny);
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Simple \t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_idriven(int nf1, int nf2, int fw_width, gpuComplex* d_fw, int M, FLT *d_kx,
		FLT *d_ky, gpuComplex *d_c, spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	hipEventRecord(start);
	if(opts.Horner){
		Spread_2d_Idriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, d_fw, M, ns,
				nf1, nf2, es_c, es_beta, fw_width);
	}else{
		Spread_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, d_fw, M, ns,
				nf1, nf2, es_c, es_beta, fw_width);
	}

#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Idriven \t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_idriven_sorted(int nf1, int nf2, int fw_width, gpuComplex* d_fw, 
		int M, FLT *d_kx, FLT *d_ky, gpuComplex *d_c, 
		spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;

	FLT *d_kxsorted,*d_kysorted;
	gpuComplex *d_csorted;

	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int numbins[2];
	int *d_binsize;
	int *d_binstartpts;
	int *d_sortidx;
	void*d_temp_storage=NULL;

	hipEventRecord(start);
	checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));
	checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
	checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1])*sizeof(int)));

#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocating GPU memory for sorted array \t%.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
			numbins[0],numbins[1],d_binsize,
			d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	int n=numbins[0]*numbins[1];
	size_t temp_storage_bytes = 0;
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts+1, n));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	PtsRearrage_noghost_2d<<<(M+1024-1)/1024,1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
			numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted,
			d_ky, d_kysorted, d_c, d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel PtsRearrange_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_kx,d_kxsorted,M*sizeof(FLT),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_ky,d_kysorted,M*sizeof(FLT),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_c,d_csorted,M*sizeof(gpuComplex),hipMemcpyDeviceToHost));
	for(int i=0; i<M; i++){
		printf("sortidx = %d, (x,y) = (%.3g, %.3g), c=(%f, %f)\n", h_sortidx[i], h_kx[i], h_ky[i], h_c[i].real(), h_c[i].imag());
	}
#endif 
	hipEventRecord(start);
	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	Spread_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kxsorted, d_kysorted, d_csorted, d_fw, M, ns,
			nf1, nf2, es_c, es_beta, fw_width);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Idriven \t\t%.3g ms\n", milliseconds);
#endif
	// Free memory
	hipEventRecord(start);
	hipFree(d_kxsorted);
	hipFree(d_kysorted);
	hipFree(d_csorted);
	hipFree(d_sortidx);
	hipFree(d_binsize);
	hipFree(d_binstartpts);
	hipFree(d_temp_storage);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree part GPU-memory \t\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_hybrid(int nf1, int nf2, int fw_width, gpuComplex* d_fw, 
		int M, FLT *d_kx, FLT *d_ky, gpuComplex *d_c, 
		spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;

	// Parameter setting
	int numbins[2];

	int *d_binsize;
	int *d_binstartpts;
	int *d_sortidx;

	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
	// assume that bin_size_x > ns/2;
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif
	FLT *d_kxsorted,*d_kysorted;
	gpuComplex *d_csorted;


	hipEventRecord(start);
	checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));

	checkCudaErrors(hipMalloc(&d_binsize,numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocating GPU memory for sorted array \t%.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
			numbins[0],numbins[1],d_binsize,
			d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize;// For debug
	h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binsize);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
#if 0
	int n=numbins[0]*numbins[1];
	int scanblocksize=1024;
	int numscanblocks=ceil((double)n/scanblocksize);
	int* d_scanblocksum, *d_scanblockstartpts;
#ifdef DEBUG
	printf("[debug ] n=%d, numscanblocks=%d\n",n,numscanblocks);
#endif 
	checkCudaErrors(hipMalloc(&d_scanblocksum,numscanblocks*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_scanblockstartpts,(numscanblocks+1)*sizeof(int)));

	for(int i=0;i<numscanblocks;i++){
		int nelemtoscan=(n-scanblocksize*i)>scanblocksize ? scanblocksize : n-scanblocksize*i;
		prescan<<<1, scanblocksize/2>>>(nelemtoscan,d_binsize+i*scanblocksize,
				d_binstartpts+i*scanblocksize,d_scanblocksum+i);
	}
#ifdef DEBUG
	int* h_scanblocksum;
	h_scanblocksum     =(int*) malloc(numscanblocks*sizeof(int));
	checkCudaErrors(hipMemcpy(h_scanblocksum,d_scanblocksum,numscanblocks*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int i=0;i<numscanblocks;i++){
		cout<<"[debug ] scanblocksum["<<i<<"]="<<h_scanblocksum[i]<<endl;
	}
#endif
	int next = pow(2, ceil(log(numscanblocks+1)/log(2)));
	if(next > 2048){
		cout<<"error: number of elements to sort exceed the prescan capability"<<endl;
		return 1;
	}
	prescan<<<1, next/2>>>(numscanblocks,d_scanblocksum,d_scanblockstartpts,d_scanblockstartpts+numscanblocks);
#ifdef DEBUG
	int* h_scanblockstartpts = (int*) malloc((numscanblocks+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_scanblockstartpts,d_scanblockstartpts,(numscanblocks+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int i=0;i<numscanblocks+1;i++){
		cout<<"[debug ] scanblockstartpts["<<i<<"]="<<h_scanblockstartpts[i]<<endl;
	}
#endif
	uniformUpdate<<<numscanblocks,scanblocksize>>>(n,d_binstartpts,d_scanblockstartpts);
#endif
	int n=numbins[0]*numbins[1];
	void *d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts+1, n));
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes)); // Allocate temporary storage for inclusive prefix scan
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts+1, n));
	checkCudaErrors(hipMemset(d_binstartpts,0,sizeof(int)));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc((numbins[0]*numbins[1]+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_binstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	cout<<"[debug ] Total number of nonuniform pts (include those in ghost bins) = "
		<< setw(4)<<h_binstartpts[numbins[0]*numbins[1]]<<endl;
	free(h_binstartpts);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	PtsRearrage_noghost_2d<<<(M+1024-1)/1024,1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
			numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted,
			d_ky, d_kysorted, d_c, d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel PtsRearrange_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	FLT *h_kxsorted, *h_kysorted;
	CPX *h_csorted;
	h_kxsorted = (FLT*)malloc(M*sizeof(FLT));
	h_kysorted = (FLT*)malloc(M*sizeof(FLT));
	h_csorted  = (CPX*)malloc(M*sizeof(CPX));
	checkCudaErrors(hipMemcpy(h_kxsorted,d_kxsorted,M*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_kysorted,d_kysorted,M*sizeof(FLT),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_csorted,d_csorted,M*sizeof(CPX),
				hipMemcpyDeviceToHost));
	for (int i=0; i<10; i++){
		cout <<"[debug ] (x,y) = ("<<setw(10)<<h_kxsorted[i]<<","
			<<setw(10)<<h_kysorted[i]<<"), bin# =  "
			<<(floor(h_kxsorted[i]/bin_size_x))+numbins[0]*(floor(h_kysorted[i]/bin_size_y))<<endl;
	}
	free(h_kysorted);
	free(h_kxsorted);
	free(h_csorted);
#endif

	hipEventRecord(start);
	threadsPerBlock.x = 16;
	threadsPerBlock.y = 16;
	blocks.x = numbins[0];
	blocks.y = numbins[1];
	size_t sharedmemorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(gpuComplex);
	if(sharedmemorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}
	// blockSize must be a multiple of bin_size_x
	Spread_2d_Hybrid<<<blocks, threadsPerBlock, sharedmemorysize>>>(d_kxsorted, d_kysorted, d_csorted, 
			d_fw, M, ns, nf1, nf2, 
			es_c, es_beta, fw_width, 
			d_binstartpts, d_binsize, 
			bin_size_x, bin_size_y);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Hybrid \t\t%.3g ms\n", milliseconds);
#endif
	// Free memory
	hipFree(d_temp_storage);
	hipEventRecord(start);
	hipFree(d_binsize);
	hipFree(d_binstartpts);
	hipFree(d_sortidx);
	hipFree(d_kxsorted);
	hipFree(d_kysorted);
	hipFree(d_csorted);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree part GPU-memory \t\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cnufftspread2d_gpu_subprob(int nf1, int nf2, int fw_width, gpuComplex* d_fw, 
		int M, FLT *d_kx, FLT *d_ky, gpuComplex *d_c, 
		spread_opts opts)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int maxsubprobsize=opts.maxsubprobsize;

	// Parameter setting
	int numbins[2];

	int *d_binsize, *d_binstartpts;
	int *d_sortidx;
	int *d_numsubprob, *d_subprobstartpts;

	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
	// assume that bin_size_x > ns/2;
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif
	FLT *d_kxsorted,*d_kysorted;
	gpuComplex *d_csorted;
	int* d_idxnupts;


	hipEventRecord(start);
	if(!opts.indirect){
		checkCudaErrors(hipMalloc(&d_kxsorted,M*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_kysorted,M*sizeof(FLT)));
		checkCudaErrors(hipMalloc(&d_csorted,M*sizeof(gpuComplex)));
	}else{
		checkCudaErrors(hipMalloc(&d_idxnupts,M*sizeof(int)));
	}
	checkCudaErrors(hipMalloc(&d_sortidx,M*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_numsubprob,  numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binsize,     numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_binstartpts, numbins[0]*numbins[1]*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_subprobstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocating GPU memory for sorted array \t%.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
			numbins[0],numbins[1],d_binsize,
			d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize;// For debug
	h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binsize);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	int n=numbins[0]*numbins[1];
	void *d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes)); // Allocate temporary storage for inclusive prefix scan
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc((numbins[0]*numbins[1])*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*numbins[1])*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_binstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binstartpts);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	if(!opts.indirect){
		hipEventRecord(start);
		PtsRearrage_noghost_2d<<<(M+1024-1)/1024,1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
				numbins[1], d_binstartpts, d_sortidx, d_kx, d_kxsorted,
				d_ky, d_kysorted, d_c, d_csorted);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel PtsRearrange_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
		FLT *h_kxsorted, *h_kysorted;
		CPX *h_csorted;
		h_kxsorted = (FLT*)malloc(M*sizeof(FLT));
		h_kysorted = (FLT*)malloc(M*sizeof(FLT));
		h_csorted  = (CPX*)malloc(M*sizeof(CPX));
		checkCudaErrors(hipMemcpy(h_kxsorted,d_kxsorted,M*sizeof(FLT),
					hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_kysorted,d_kysorted,M*sizeof(FLT),
					hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(h_csorted,d_csorted,M*sizeof(CPX),
					hipMemcpyDeviceToHost));
		for (int i=0; i<10; i++){
			cout <<"[debug ] (x,y) = ("<<setw(10)<<h_kxsorted[i]<<","
				<<setw(10)<<h_kysorted[i]<<"), bin# =  "
				<<(floor(h_kxsorted[i]/bin_size_x))+numbins[0]*(floor(h_kysorted[i]/bin_size_y))<<endl;
		}
		free(h_kysorted);
		free(h_kxsorted);
		free(h_csorted);
#endif
	}else{
		hipEventRecord(start);
		CalcInvertofGlobalSortIdx_2d<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,bin_size_y,numbins[0],
				numbins[1],d_binstartpts,d_sortidx,
				d_kx,d_ky,d_idxnupts);
#ifdef DEBUG
		int *h_idxnupts;
		h_idxnupts = (int*)malloc(M*sizeof(int));
		checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,M*sizeof(int),hipMemcpyDeviceToHost));
		for (int i=0; i<M; i++){
			cout <<"[debug ] idx="<< h_idxnupts[i]<<endl;
		}
		free(h_idxnupts);
#endif
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel CalcInvertofGlobalSortIdx_2d \t%.3g ms\n", milliseconds);
#endif
	}

	/* --------------------------------------------- */
	//        Determining Subproblem properties      //
	/* --------------------------------------------- */
	hipEventRecord(start);
	CalcSubProb_2d<<<(M+1024-1)/1024, 1024>>>(d_binsize,d_numsubprob,maxsubprobsize,numbins[0]*numbins[1]);
#ifdef DEBUG
	int* h_numsubprob;
	h_numsubprob = (int*) malloc(n*sizeof(int));
	checkCudaErrors(hipMemcpy(h_numsubprob,d_numsubprob,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_numsubprob[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_numsubprob);
#endif
	// Scanning the same length array, so we don't need calculate temp_storage_bytes here
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_numsubprob, d_subprobstartpts+1, n));
	checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));

#ifdef DEBUG
	printf("[debug ] Subproblem start points\n");
	int* h_subprobstartpts;
	h_subprobstartpts = (int*) malloc((n+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprobstartpts,d_subprobstartpts,(n+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_subprobstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
	free(h_subprobstartpts);
#endif

	int totalnumsubprob;
	checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],sizeof(int),
				hipMemcpyDeviceToHost));
	int* d_subprob_to_bin;
	checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
	MapBintoSubProb_2d<<<(numbins[0]*numbins[1]+1024-1)/1024, 1024>>>(d_subprob_to_bin, 
			d_subprobstartpts,
			d_numsubprob,
			numbins[0]*numbins[1]);
#ifdef DEBUG
	printf("[debug ] Map Subproblem to Bins\n");
	int* h_subprob_to_bin;
	h_subprob_to_bin = (int*) malloc((totalnumsubprob)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprob_to_bin,d_subprob_to_bin,(totalnumsubprob)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<totalnumsubprob; j++){
		cout<<"[debug ] ";
		cout <<"nsub["<<j<<"] = "<<setw(2)<<h_subprob_to_bin[j];
		cout<<endl;
	}
	free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif

	hipEventRecord(start);
	size_t sharedmemorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(gpuComplex);
	if(sharedmemorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	if(!opts.indirect){
		// blockSize must be a multiple of bin_size_x
		Spread_2d_Subprob<<<totalnumsubprob, 256, sharedmemorysize>>>(d_kxsorted, d_kysorted, d_csorted, 
				d_fw, M, ns, nf1, nf2, 
				es_c, es_beta, fw_width, 
				d_binstartpts, d_binsize, 
				bin_size_x, bin_size_y,
				d_subprob_to_bin, d_subprobstartpts, 
				d_numsubprob, maxsubprobsize, 
				numbins[0], numbins[1]);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel Spread_2d_Subprob \t\t%.3g ms\n", milliseconds);
#endif
	}else{
		Spread_2d_Subprob_V2<<<totalnumsubprob, 256, sharedmemorysize>>>(d_kx, d_ky, d_c,
				d_fw, M, ns, nf1, nf2,
				es_c, es_beta, fw_width,
				d_binstartpts, d_binsize,
				bin_size_x, bin_size_y,
				d_subprob_to_bin, d_subprobstartpts,
				d_numsubprob, maxsubprobsize,
				numbins[0], numbins[1], d_idxnupts);
#ifdef SPREADTIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tKernel Spread_2d_Subprob_V2 \t\t%.3g ms\n", milliseconds);
#endif
	}
	hipEventRecord(start);
	// Free memory
	hipFree(d_temp_storage);
	hipFree(d_binsize);
	hipFree(d_binstartpts);
	hipFree(d_sortidx);
	hipFree(d_numsubprob);
	hipFree(d_subprobstartpts);
	hipFree(d_subprob_to_bin);
	if(!opts.indirect){
		hipFree(d_kxsorted);
		hipFree(d_kysorted);
		hipFree(d_csorted);
	}else{
		hipFree(d_idxnupts);
	}
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree part GPU-memory \t\t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}
