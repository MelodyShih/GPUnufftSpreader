#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "spreadinterp.h"

using namespace std;

int allocgpumem2d_plan(cufinufft_plan *d_plan)
{
	int ms = d_plan->ms;
	int mt = d_plan->mt;
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int ntransfcufftplan = d_plan->ntransfcufftplan;

	d_plan->byte_now=0;
	// No extra memory is needed in idriven method (case 1)
	switch(d_plan->opts.gpu_method)
	{
		case 5:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/d_plan->opts.gpu_binsizex);
				numbins[1] = ceil((FLT) nf2/d_plan->opts.gpu_binsizey);
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
		case 6:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/d_plan->opts.gpu_binsizex);
				numbins[1] = ceil((FLT) nf2/d_plan->opts.gpu_binsizey);
				checkCudaErrors(hipMalloc(&d_plan->finegridsize,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->fgstartpts,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
	}

	checkCudaErrors(hipMalloc(&d_plan->fw, ntransfcufftplan*nf1*nf2*
			sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_plan->fk,ntransfcufftplan*ms*mt*
		sizeof(CUCPX)));

	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));

	hipStream_t* streams =(hipStream_t*) malloc(d_plan->opts.gpu_nstreams*
		sizeof(hipStream_t));
	for(int i=0; i<d_plan->opts.gpu_nstreams; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));
	d_plan->streams = streams;
	return 0;
}

int allocgpumem2d_nupts(cufinufft_plan *d_plan)
{
	int M = d_plan->M;
	int ntransfcufftplan = d_plan->ntransfcufftplan;

	checkCudaErrors(hipMalloc(&d_plan->kx,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->ky,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->c,ntransfcufftplan*M*sizeof(CUCPX)));
	switch(d_plan->opts.gpu_method)
	{
		case 5:
		case 6:
			{
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
			}
			break;
	}
	return 0;
}

void freegpumemory2d(cufinufft_plan *d_plan)
{
	hipFree(d_plan->fw);
	hipFree(d_plan->kx);
	hipFree(d_plan->ky);
	hipFree(d_plan->c);
	hipFree(d_plan->fwkerhalf1);
	hipFree(d_plan->fwkerhalf2);
	switch(d_plan->opts.gpu_method)
	{
		case 5:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case 6:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->finegridsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for(int i=0; i<d_plan->opts.gpu_nstreams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));
}

int allocgpumemory3d(const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	int ms = d_plan->ms;
	int mt = d_plan->mt;
	int mu = d_plan->mu;
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int nf3 = d_plan->nf3;
	int M = d_plan->M;
	int ntransfcufftplan = d_plan->ntransfcufftplan;

	d_plan->byte_now=0;
	// No extra memory is needed in idriven method;
	switch(opts.method)
	{
		case 6:
			{
				int numobins[3], numbins[3];
				numobins[0] = ceil((FLT) nf1/opts.o_bin_size_x);
				numobins[1] = ceil((FLT) nf2/opts.o_bin_size_y);
				numobins[2] = ceil((FLT) nf3/opts.o_bin_size_z);

				numbins[0] = numobins[0]*opts.o_bin_size_x/opts.bin_size_x;
				numbins[1] = numobins[1]*opts.o_bin_size_y/opts.bin_size_y;
				numbins[2] = numobins[2]*opts.o_bin_size_z/opts.bin_size_z;
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,
					numobins[0]*numobins[1]*numobins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numnupts,
					numobins[0]*numobins[1]*numobins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,
					numbins[0]*numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,
					numbins[0]*numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,(numobins[0]
					*numobins[1]*numobins[2]+1)*sizeof(int)));
			}
			break;
		case 1:
		case 2:
		case 3:
			{
				int numobins[3], numbins[3];
				int binsperobins[3];
				numobins[0] = ceil((FLT) nf1/opts.o_bin_size_x);
				numobins[1] = ceil((FLT) nf2/opts.o_bin_size_y);
				numobins[2] = ceil((FLT) nf3/opts.o_bin_size_z);

				binsperobins[0] = opts.o_bin_size_x/opts.bin_size_x;
				binsperobins[1] = opts.o_bin_size_y/opts.bin_size_y;
				binsperobins[2] = opts.o_bin_size_z/opts.bin_size_z;

				numbins[0] = numobins[0]*(binsperobins[0]+2);
				numbins[1] = numobins[1]*(binsperobins[1]+2);
				numbins[2] = numobins[2]*(binsperobins[2]+2);

				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,
					numobins[0]*numobins[1]*numobins[2]*sizeof(int)));
				//checkCudaErrors(hipMalloc(&d_plan->numnupts,
					//numobins[0]*numobins[1]*numobins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,
					numbins[0]*numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,
					(numbins[0]*numbins[1]*numbins[2]+1)*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,(numobins[0]
					*numobins[1]*numobins[2]+1)*sizeof(int)));
			}
			break;
		case 4:
			{
				if(opts.sort){
					int numbins[3];
					numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
					numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
					numbins[2] = ceil((FLT) nf3/opts.bin_size_z);
					checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*numbins[2]*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*numbins[2]*sizeof(int)));
					checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*numbins[2]*sizeof(int)));
				}else{
					checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				}
			}
			break;
		case 5:
			{
				int numbins[3];
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				numbins[2] = ceil((FLT) nf3/opts.bin_size_z);
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
					numbins[1]*numbins[2]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
					(numbins[0]*numbins[1]*numbins[2]+1)*sizeof(int)));
			}
			break;
	}
	checkCudaErrors(hipMalloc(&d_plan->kx,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->ky,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->kz,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->c,ntransfcufftplan*M*sizeof(CUCPX)));

	//size_t pitch;
	checkCudaErrors(hipMalloc(&d_plan->fw, ntransfcufftplan*nf1*nf2*nf3*
		sizeof(CUCPX)));

	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf3,(nf3/2+1)*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->fk,ntransfcufftplan*ms*mt*mu*
		sizeof(CUCPX)));

	d_plan->nstreams=min(16, d_plan->ntransfcufftplan);
	hipStream_t* streams =(hipStream_t*) malloc(d_plan->nstreams*
		sizeof(hipStream_t));
	for(int i=0; i<d_plan->nstreams; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));
	d_plan->streams = streams;

	return 0;
}
void freegpumemory3d(const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	hipFree(d_plan->fw);
	hipFree(d_plan->kx);
	hipFree(d_plan->ky);
	hipFree(d_plan->kz);
	hipFree(d_plan->c);
	hipFree(d_plan->fwkerhalf1);
	hipFree(d_plan->fwkerhalf2);
	hipFree(d_plan->fwkerhalf3);
	switch(opts.method)
	{
		case 6:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
				checkCudaErrors(hipFree(d_plan->subprob_to_nupts));
			}
			break;
		case 1:
		case 2:
		case 3:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case 4:
			{
				if(opts.sort){
					checkCudaErrors(hipFree(d_plan->idxnupts));
					checkCudaErrors(hipFree(d_plan->sortidx));
					checkCudaErrors(hipFree(d_plan->binsize));
					checkCudaErrors(hipFree(d_plan->binstartpts));
				}else{
					checkCudaErrors(hipFree(d_plan->idxnupts));
				}
			}
			break;
		case 5:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->temp_storage));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}
	for(int i=0; i<d_plan->nstreams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));
}

int allocgpumem1d_plan(cufinufft_plan *d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
int allocgpumem1d_nupts(cufinufft_plan *d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
void freegpumemory1d(cufinufft_plan *d_plan)
{
	cerr<<"Not yet implemented"<<endl;
}

int allocgpumem3d_plan(cufinufft_plan *d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
int allocgpumem3d_nupts(cufinufft_plan *d_plan)
{
	cerr<<"Not yet implemented"<<endl;
	return 1;
}
void freegpumemory3d(cufinufft_plan *d_plan)
{
	cerr<<"Not yet implemented"<<endl;
}
