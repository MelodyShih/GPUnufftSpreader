#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "spreadinterp.h"

using namespace std;

int allocgpumemory(const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	int ms = d_plan->ms;
	int mt = d_plan->mt;
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int M = d_plan->M;

	d_plan->byte_now=0;
	// No extra memory is needed in idriven method;
	switch(opts.method)
	{
		case 2:
			{
				//int total_mem_in_bytes=
				checkCudaErrors(hipMalloc(&d_plan->kxsorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_plan->kysorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_plan->csorted,M*sizeof(CUCPX)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));

				int numbins[2];
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,(numbins[0]*numbins[1])*sizeof(int)));
			}
			break;
		case 4:
			{
				checkCudaErrors(hipMalloc(&d_plan->kxsorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_plan->kysorted,M*sizeof(FLT)));
				checkCudaErrors(hipMalloc(&d_plan->csorted,M*sizeof(CUCPX)));

				int numbins[2];
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
		case 5:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,  numbins[0]*numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,     numbins[0]*numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts, numbins[0]*numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
	}
	checkCudaErrors(hipMalloc(&d_plan->kx,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->ky,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->c,M*sizeof(CUCPX)));

	size_t pitch;
	checkCudaErrors(hipMallocPitch((void**) &d_plan->fw, &pitch,nf1*sizeof(CUCPX),nf2));
	d_plan->fw_width = pitch/sizeof(CUCPX);

	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->fk,ms*mt*sizeof(CUCPX)));

	return 0;
}
void freegpumemory(const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	hipFree(d_plan->fw);
	hipFree(d_plan->kx);
	hipFree(d_plan->ky);
	hipFree(d_plan->c);
	hipFree(d_plan->fwkerhalf1);
	hipFree(d_plan->fwkerhalf2);
	switch(opts.method)
	{
		case 2:
			{
				checkCudaErrors(hipFree(d_plan->kxsorted));
				checkCudaErrors(hipFree(d_plan->kysorted));
				checkCudaErrors(hipFree(d_plan->csorted));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->temp_storage));
			}
			break;
		case 4:
			{
				checkCudaErrors(hipFree(d_plan->kxsorted));
				checkCudaErrors(hipFree(d_plan->kysorted));
				checkCudaErrors(hipFree(d_plan->csorted));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->temp_storage));
			}
			break;
		case 5:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->temp_storage));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}
}
