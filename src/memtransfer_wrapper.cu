#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "spreadinterp.h"

using namespace std;

int allocgpumemory2d(const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	int ms = d_plan->ms;
	int mt = d_plan->mt;
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int M = d_plan->M;
	int ntransfcufftplan = d_plan->ntransfcufftplan;

	d_plan->byte_now=0;
	// No extra memory is needed in idriven method (case 1);
	switch(opts.method)
	{
		case 5:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx, M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
		case 6:
			{
				int numbins[2];
				numbins[0] = ceil((FLT) nf1/opts.bin_size_x);
				numbins[1] = ceil((FLT) nf2/opts.bin_size_y);
				checkCudaErrors(hipMalloc(&d_plan->finegridsize,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->fgstartpts,nf1*nf2*
						sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->idxnupts,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->sortidx,M*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->numsubprob,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binsize,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->binstartpts,numbins[0]*
						numbins[1]*sizeof(int)));
				checkCudaErrors(hipMalloc(&d_plan->subprobstartpts,
						(numbins[0]*numbins[1]+1)*sizeof(int)));
			}
			break;
	}
	checkCudaErrors(hipMalloc(&d_plan->kx,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->ky,M*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->c,ntransfcufftplan*M*sizeof(CUCPX)));

	checkCudaErrors(hipMalloc(&d_plan->fw, ntransfcufftplan*nf1*nf2*
			sizeof(CUCPX)));
	checkCudaErrors(hipMalloc(&d_plan->fk,ntransfcufftplan*ms*mt*
		sizeof(CUCPX)));

	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf1,(nf1/2+1)*sizeof(FLT)));
	checkCudaErrors(hipMalloc(&d_plan->fwkerhalf2,(nf2/2+1)*sizeof(FLT)));

	hipStream_t* streams =(hipStream_t*) malloc(opts.nstreams*
		sizeof(hipStream_t));
	for(int i=0; i<opts.nstreams; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));
	d_plan->streams = streams;

	return 0;
}
void freegpumemory2d(const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	hipFree(d_plan->fw);
	hipFree(d_plan->kx);
	hipFree(d_plan->ky);
	hipFree(d_plan->c);
	hipFree(d_plan->fwkerhalf1);
	hipFree(d_plan->fwkerhalf2);
	switch(opts.method)
	{
		case 5:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->temp_storage));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
		case 6:
			{
				checkCudaErrors(hipFree(d_plan->idxnupts));
				checkCudaErrors(hipFree(d_plan->sortidx));
				checkCudaErrors(hipFree(d_plan->numsubprob));
				checkCudaErrors(hipFree(d_plan->binsize));
				checkCudaErrors(hipFree(d_plan->finegridsize));
				checkCudaErrors(hipFree(d_plan->binstartpts));
				checkCudaErrors(hipFree(d_plan->subprobstartpts));
				checkCudaErrors(hipFree(d_plan->temp_storage));
				checkCudaErrors(hipFree(d_plan->subprob_to_bin));
			}
			break;
	}

	for(int i=0; i<opts.nstreams; i++)
		checkCudaErrors(hipStreamDestroy(d_plan->streams[i]));
}
