#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "spread.h"
#include "memtransfer.h"

using namespace std;

// This function includes device memory allocation, transfer, free
int cufinufft_interp2d(int ms, int mt, int nf1, int nf2, CPX* h_fw, int M, FLT *h_kx,
		FLT *h_ky, CPX *h_c, spread_opts opts, cufinufft_plan* d_plan)
{
	if(opts.spread_direction!=2){
		printf("spread direction not set\n");
		return 1;
	}
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	
	d_plan->ms = ms;
        d_plan->mt = mt;
        d_plan->nf1 = nf1;
        d_plan->nf2 = nf2;
	d_plan->M = M;

        d_plan->h_kx = h_kx;
        d_plan->h_ky = h_ky;
        d_plan->h_c = h_c;
	d_plan->h_fw = h_fw;
	d_plan->h_fwkerhalf1 = NULL;
	d_plan->h_fwkerhalf2 = NULL;

	if(opts.pirange){
		for(int i=0; i<M; i++){
			h_kx[i]=RESCALE(h_kx[i], nf1, opts.pirange);
			h_ky[i]=RESCALE(h_ky[i], nf2, opts.pirange);
		}
	}
	hipEventRecord(start);
	ier = allocgpumemory(opts, d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Allocate GPU memory\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	ier = copycpumem_to_gpumem(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory HtoD\t %.3g ms\n", milliseconds);
#endif
	ier = cuinterp2d(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Spread\t\t\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	ier = copygpumem_to_cpumem_c(d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory DtoH\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	free_gpumemory(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
	return ier;
}

// a wrapper of different methods of spreader
int cuinterp2d( spread_opts opts, cufinufft_plan* d_plan)
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int fw_width = d_plan->fw_width;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch(opts.method)
	{
		case 1:
			{
				hipEventRecord(start);
				ier = cuinterp2d_idriven(nf1, nf2, fw_width, M, opts, d_plan);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_idriven"<<endl;
					return 1;
				}
			}
			break;
		case 5:
			{
				hipEventRecord(start);
				ier = cuinterp2d_subprob(nf1, nf2, fw_width, M, opts, d_plan);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_hybrid"<<endl;
					return 1;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1 or 5"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Interp " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}

int cuinterp2d_idriven(int nf1, int nf2, int fw_width, int M, spread_opts opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	hipEventRecord(start);
	Interp_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, d_fw, M, ns,
						       nf1, nf2, es_c, es_beta, fw_width);

#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_2d_Idriven \t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cuinterp2d_subprob(int nf1, int nf2, int fw_width, int M, spread_opts opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int maxsubprobsize=opts.maxsubprobsize;

	// assume that bin_size_x > ns/2;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int numbins[2];
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_sortidx = d_plan->sortidx;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;
	d_plan->subprob_to_bin = NULL;
	int *d_subprob_to_bin = d_plan->subprob_to_bin;
	d_plan->temp_storage = NULL;
	void *d_temp_storage = d_plan->temp_storage;

	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,bin_size_y,
			numbins[0],numbins[1],d_binsize,
			d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize;// For debug
	h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binsize);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	int n=numbins[0]*numbins[1];
	size_t temp_storage_bytes = 0;
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes)); // Allocate temporary storage for inclusive prefix scan
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_binsize, d_binstartpts, n));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc((numbins[0]*numbins[1])*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,(numbins[0]*numbins[1])*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_binstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binstartpts);
	cout<<"[debug ] --------------------------------------------------------------"<<endl;
#endif

	hipEventRecord(start);
	CalcInvertofGlobalSortIdx_2d<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,bin_size_y,numbins[0],
			numbins[1],d_binstartpts,d_sortidx,
			d_kx,d_ky,d_idxnupts);
#ifdef DEBUG
	int *h_idxnupts;
	h_idxnupts = (int*)malloc(M*sizeof(int));
	checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,M*sizeof(int),hipMemcpyDeviceToHost));
	for (int i=0; i<M; i++){
		cout <<"[debug ] idx="<< h_idxnupts[i]<<endl;
	}
	free(h_idxnupts);
#endif
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcInvertofGlobalSortIdx_2d \t%.3g ms\n", milliseconds);
#endif

	/* --------------------------------------------- */
	//        Determining Subproblem properties      //
	/* --------------------------------------------- */
	hipEventRecord(start);
	CalcSubProb_2d<<<(M+1024-1)/1024, 1024>>>(d_binsize,d_numsubprob,maxsubprobsize,numbins[0]*numbins[1]);
#ifdef DEBUG
	int* h_numsubprob;
	h_numsubprob = (int*) malloc(n*sizeof(int));
	checkCudaErrors(hipMemcpy(h_numsubprob,d_numsubprob,numbins[0]*numbins[1]*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_numsubprob[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_numsubprob);
#endif
	// Scanning the same length array, so we don't need calculate temp_storage_bytes here
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_numsubprob, d_subprobstartpts+1, n));
	checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));

#ifdef DEBUG
	printf("[debug ] Subproblem start points\n");
	int* h_subprobstartpts;
	h_subprobstartpts = (int*) malloc((n+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprobstartpts,d_subprobstartpts,(n+1)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<h_subprobstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
	free(h_subprobstartpts);
#endif

	int totalnumsubprob;
	checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],sizeof(int),
				hipMemcpyDeviceToHost));
	checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
	MapBintoSubProb_2d<<<(numbins[0]*numbins[1]+1024-1)/1024, 1024>>>(d_subprob_to_bin, 
			d_subprobstartpts,
			d_numsubprob,
			numbins[0]*numbins[1]);
#ifdef DEBUG
	printf("[debug ] Map Subproblem to Bins\n");
	int* h_subprob_to_bin;
	h_subprob_to_bin = (int*) malloc((totalnumsubprob)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprob_to_bin,d_subprob_to_bin,(totalnumsubprob)*sizeof(int),
				hipMemcpyDeviceToHost));
	for(int j=0; j<totalnumsubprob; j++){
		cout<<"[debug ] ";
		cout <<"nsub["<<j<<"] = "<<setw(2)<<h_subprob_to_bin[j];
		cout<<endl;
	}
	free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
	FLT sigma=opts.upsampfac;
	hipEventRecord(start);
	size_t sharedplanorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(CUCPX);
	if(sharedplanorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	Interp_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(d_kx, d_ky, d_c,
			d_fw, M, ns, nf1, nf2,
			es_c, es_beta, sigma, fw_width,
			d_binstartpts, d_binsize,
			bin_size_x, bin_size_y,
			d_subprob_to_bin, d_subprobstartpts,
			d_numsubprob, maxsubprobsize,
			numbins[0], numbins[1], d_idxnupts);
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_2d_Subprob_V2 \t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}
