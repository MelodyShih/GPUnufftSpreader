#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include "spread.h"
#include "memtransfer.h"
#include "deconvolve.h"
#include "cufinufft.h"
#include "finufft/utils.h"
#include "finufft/common.h"

using namespace std;

int cufinufft2d1_plan(int M, FLT* h_kx, FLT* h_ky, CPX* h_c, int ms, int mt, CPX* h_fk, 
		int iflag, FLT eps, FLT upsampfac, spread_opts &opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier=setup_cuspreader(opts,eps,upsampfac);
	int nf1 = (int) opts.upsampfac*ms;
	int nf2 = (int) opts.upsampfac*mt;
	int fftsign = (iflag>=0) ? 1 : -1;
	
	d_plan->ms = ms;
	d_plan->mt = mt;
	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->h_kx = h_kx;
	d_plan->h_ky = h_ky;
	d_plan->h_c = h_c;
	d_plan->h_fk = h_fk;
	d_plan->M = M;
	d_plan->iflag = fftsign; 
#ifdef INFO
	printf("[info  ] 2d1: (ms,mt)=(%d,%d) (nf1, nf2)=(%d,%d) nj=%d\n", ms, mt, d_plan->nf1, d_plan->nf2, d_plan->M);
#endif

	// this may move to gpu
	CNTime timer; timer.start();
	FLT *fwkerhalf1 = (FLT*)malloc(sizeof(FLT)*(nf1/2+1));
	FLT *fwkerhalf2 = (FLT*)malloc(sizeof(FLT)*(nf2/2+1));
	onedim_fseries_kernel(nf1, fwkerhalf1, opts);
	onedim_fseries_kernel(nf2, fwkerhalf2, opts);
	d_plan->h_fwkerhalf1 = fwkerhalf1;
	d_plan->h_fwkerhalf2 = fwkerhalf2;
#ifdef TIME
	printf("[time  ] \tkernel fser (ns=%d):\t %.3g s\n", opts.nspread,timer.elapsedsec());
#endif

	hipEventRecord(start);
	ier = allocgpumemory(opts, d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tAllocate GPU memory\t %.3g s\n", milliseconds/1000);
#endif
	hipEventRecord(start);
	hipfftHandle fftplan;
	int ndata=1;
	int n[] = {nf2, nf1};
	int inembed[] = {nf2, d_plan->fw_width};
	hipfftPlanMany(&fftplan,2,n,inembed,1,inembed[0]*inembed[1],inembed,1,inembed[0]*inembed[1],
			CUFFT_TYPE,ndata);
	d_plan->fftplan = fftplan;
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tCUFFT Plan\t\t %.3g s\n", milliseconds/1000);
#endif
	return ier;
}

int cufinufft2d1_exec(spread_opts opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	if(opts.pirange){
		for(int i=0; i<d_plan->M; i++){
			d_plan->h_kx[i]=RESCALE(d_plan->h_kx[i], d_plan->nf1, opts.pirange);
			d_plan->h_ky[i]=RESCALE(d_plan->h_ky[i], d_plan->nf2, opts.pirange);
		}
	}	

	// Copy memory to device
	int ier = copycpumem_to_gpumem(d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tCopy memory HtoD\t %.3g s\n", milliseconds/1000);
#endif
	// Step 1: Spread
        hipEventRecord(start);
        ier = cuspread2d(opts, d_plan);
        if(ier != 0 ){
                printf("error: cuspread2d, method(%d)\n", opts.method);
                return 0;
        }
#ifdef TIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tSpread\t\t\t %.3g s\n", milliseconds/1000);
#endif
	// Step 2: FFT
	hipEventRecord(start);
	CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif

	// Step 3: deconvolve and shuffle
	hipEventRecord(start);
	cudeconvolve2d(opts,d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tDeconvolve\t\t %.3g s\n", milliseconds/1000);
#endif
	hipEventRecord(start);
	ier = copygpumem_to_cpumem_fk(d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tCopy memory DtoH\t %.3g s\n", milliseconds/1000);
#endif
	return ier;
}

int cufinufft2d1_destroy(spread_opts opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
        hipfftDestroy(d_plan->fftplan);
        free_gpumemory(opts, d_plan);
	free(d_plan->h_fwkerhalf1);
	free(d_plan->h_fwkerhalf2);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tFree gpu memory\t\t %.3g s\n", milliseconds/1000);
#endif
	return 0;
}

int cufinufft2d(int ms, int mt, int M, FLT* h_kx, FLT* h_ky, CPX* h_c, FLT tol, 
		int iflag, int nf1, int nf2, CPX* h_fk, spread_opts opts, 
		cufinufft_plan* d_plan, FLT *fwkerhalf1, FLT *fwkerhalf2)
{
	d_plan->ms = ms;
	d_plan->mt = mt;
	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->M = M;
	d_plan->h_kx = h_kx;
	d_plan->h_ky = h_ky;
	d_plan->h_c = h_c;
	d_plan->h_fk = h_fk;
	d_plan->h_fwkerhalf1 = fwkerhalf1;
	d_plan->h_fwkerhalf2 = fwkerhalf2;

	if(opts.pirange){
		for(int i=0; i<M; i++){
			h_kx[i]=RESCALE(h_kx[i], nf1, opts.pirange);
			h_ky[i]=RESCALE(h_ky[i], nf2, opts.pirange);
		}
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int ier;
	// Step 0: Allocate and transfer memory for GPU
#ifdef INFO
	cout<<"[info  ] Spreading "<<M<<" pts to ["<<nf1<<"x"<<nf2<<"] uniform grids"<<endl;
#endif

	hipEventRecord(start);
	ier = allocgpumemory(opts, d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Allocate GPU memory\t %.3g s\n", milliseconds/1000);
#endif

	hipEventRecord(start);
	ier = copycpumem_to_gpumem(d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory HtoD\t %.3g s\n", milliseconds/1000);
#endif

	// Step 1: Spread
	hipEventRecord(start);
	ier = cuspread2d(opts, d_plan);
	if(ier != 0 ){
		cout<<"error: cnufftspread2d_gpu_idriven"<<endl;
		return 0;
	}
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Spread\t\t\t %.3g s\n", milliseconds/1000);
#endif
	// Step 2: Call FFT
	hipEventRecord(start);
	hipfftHandle plan;
	int ndata=1;
	int n[] = {nf2, nf1};
	int inembed[] = {nf2, d_plan->fw_width};
	hipfftPlanMany(&plan,2,n,inembed,1,inembed[0]*inembed[1],inembed,1,inembed[0]*inembed[1],
			CUFFT_TYPE,ndata);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] CUFFT Plan\t\t %.3g s\n", milliseconds/1000);
#endif
	hipEventRecord(start);
	CUFFT_EX(plan, d_plan->fw, d_plan->fw, iflag);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] CUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif
	// Step 3: deconvolve and shuffle
	hipEventRecord(start);
	cudeconvolve2d(opts,d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Deconvolve\t\t %.3g s\n", milliseconds/1000);
#endif

	hipEventRecord(start);
	ier = copygpumem_to_cpumem_fk(d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory DtoH\t %.3g s\n", milliseconds/1000);
#endif
	hipfftDestroy(plan);
	free_gpumemory(opts, d_plan);
	return 0;
}
