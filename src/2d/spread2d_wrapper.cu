#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <assert.h>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "../spreadinterp.h"
#include "../memtransfer.h"

using namespace std;

// This is a function only doing spread includes device memory allocation, transfer, free
int cufinufft_spread2d(int ms, int mt, int nf1, int nf2, CPX* h_fw, int M, 
		const FLT *h_kx, const FLT *h_ky, const CPX *h_c, 
		cufinufft_opts &opts, cufinufft_plan* d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;

	d_plan->ms = ms;
	d_plan->mt = mt;
	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->M = M;
	d_plan->ntransfcufftplan = 1;

	hipEventRecord(start);
	ier = allocgpumemory2d(opts, d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Allocate GPU memory\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(d_plan->kx,h_kx,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->ky,h_ky,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->c, h_c, M*sizeof(CUCPX),
		hipMemcpyHostToDevice));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory HtoD (%d Bytes) \t%.3g ms\n", 
		2*M*sizeof(FLT)+M*sizeof(CUCPX), milliseconds);
#endif

	if(opts.method == 5){
		ier = cuspread2d_subprob_prop(nf1,nf2,M,opts,d_plan);
		if(ier != 0 ){
			printf("error: cuspread2d_subprob_prop, method(%d)\n", opts.method);
			return 0;
		}
	}

	if(opts.method == 6){
		ier = cuspread2d_paul_prop(nf1,nf2,M,opts,d_plan);
		if(ier != 0 ){
			printf("error: cuspread2d_subprob_prop, method(%d)\n", opts.method);
			return 0;
		}
	}

	hipEventRecord(start);
	ier = cuspread2d(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Spread (%d)\t\t %.3g ms\n", opts.method, milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(h_fw,d_plan->fw,nf1*nf2*sizeof(CUCPX),
		hipMemcpyDeviceToHost));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory DtoH (%d Bytes) \t %.3g ms\n",
		nf1*nf2*sizeof(CUCPX),  milliseconds);
#endif
	hipEventRecord(start);
	freegpumemory2d(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
	return ier;
}

// a wrapper of different methods of spreader
int cuspread2d(cufinufft_opts &opts, cufinufft_plan* d_plan)
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch(opts.method)
	{
		case 1:
			{
				hipEventRecord(start);
				ier = cuspread2d_idriven(nf1, nf2, M, opts, d_plan);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_idriven"<<endl;
					return 1;
				}
			}
			break;
		case 5:
			{
				hipEventRecord(start);
				ier = cuspread2d_subprob(nf1, nf2, M, opts, d_plan);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_subprob"<<endl;
					return 1;
				}
			}
			break;
		case 6:
			{
				hipEventRecord(start);
				ier = cuspread2d_paul(nf1, nf2, M, opts, d_plan);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_paul"<<endl;
					return 1;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 5"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Spread " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}

int cuspread2d_idriven(int nf1, int nf2, int M, const cufinufft_opts opts, 
	cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	FLT sigma=opts.upsampfac;

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	hipEventRecord(start);
	if(opts.Horner){
		Spread_2d_Idriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, 
			d_fw, M, ns, nf1, nf2, sigma);
	}else{
		Spread_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c, d_fw, 
			M, ns, nf1, nf2, es_c, es_beta);
	}

#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Idriven \t%.3g ms\n", milliseconds);
#endif
	return 0;
}

// this function determines the properties for spreading that are independent
// of the strength of the nodes,  only relates to the locations of the nodes, 
// which only needs to be done once
int cuspread2d_subprob_prop(int nf1, int nf2, int M, const cufinufft_opts opts, 
	cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int maxsubprobsize=opts.maxsubprobsize;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int numbins[2];
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
#ifdef DEBUG
	cout<<"[debug  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[debug  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

	FLT*   d_kx = d_plan->kx;
	FLT*   d_ky = d_plan->ky;

#ifdef DEBUG
	FLT *h_kx;
	FLT *h_ky;
	h_kx = (FLT*)malloc(M*sizeof(FLT));
	h_ky = (FLT*)malloc(M*sizeof(FLT));

	checkCudaErrors(hipMemcpy(h_kx,d_kx,M*sizeof(FLT),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_ky,d_ky,M*sizeof(FLT),hipMemcpyDeviceToHost));
	for(int i=0; i<M; i++){
		cout<<"[debug ]";
		cout <<"("<<setw(3)<<h_kx[i]<<","<<setw(3)<<h_ky[i]<<")"<<endl;
	}
#endif
	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_sortidx = d_plan->sortidx;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;

	int *d_subprob_to_bin = NULL;

	d_plan->temp_storage = NULL;
	void *d_temp_storage = d_plan->temp_storage;

	hipEventRecord(start);
	checkCudaErrors(hipMemset(d_binsize,0,numbins[0]*numbins[1]*sizeof(int)));
	CalcBinSize_noghost_2d<<<(M+1024-1)/1024, 1024>>>(M,nf1,nf2,bin_size_x,
		bin_size_y,numbins[0],numbins[1],d_binsize,d_kx,d_ky,d_sortidx);
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", 
		milliseconds);
#endif
#ifdef DEBUG
	int *h_binsize;// For debug
	h_binsize     = (int*)malloc(numbins[0]*numbins[1]*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binsize,d_binsize,numbins[0]*numbins[1]*
		sizeof(int),hipMemcpyDeviceToHost));
	cout<<"[debug ] bin size:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<" bin["<<setw(3)<<i<<","<<setw(3)<<j<<"]="<<
				h_binsize[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binsize);
	cout<<"[debug ] ----------------------------------------------------"<<endl;
#endif
#ifdef DEBUG
	int *h_sortidx;
	h_sortidx = (int*)malloc(M*sizeof(int));
	checkCudaErrors(hipMemcpy(h_sortidx,d_sortidx,M*sizeof(int),
		hipMemcpyDeviceToHost));
	cout<<"[debug ]";
	for(int i=0; i<M; i++){
		cout <<"point["<<setw(3)<<i<<"]="<<setw(3)<<h_sortidx[i]<<endl;
	}

#endif

	hipEventRecord(start);
	int n=numbins[0]*numbins[1];
	size_t temp_storage_bytes = 0;
	assert(d_temp_storage == NULL);
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, 
				temp_storage_bytes, 
				d_binsize, d_binstartpts, 
				n));
	// Allocate temporary storage for inclusive prefix scan
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes)); 
	CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, 
				temp_storage_bytes, 
				d_binsize, d_binstartpts, 
				n));
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
	int *h_binstartpts;
	h_binstartpts = (int*)malloc((numbins[0]*numbins[1])*sizeof(int));
	checkCudaErrors(hipMemcpy(h_binstartpts,d_binstartpts,
				(numbins[0]*numbins[1])*sizeof(int),
				hipMemcpyDeviceToHost));
	cout<<"[debug ] Result of scan bin_size array:"<<endl;
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"bin["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)
				<<h_binstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_binstartpts);
	cout<<"[debug ] ---------------------------------------------------"<<endl;
#endif
	hipEventRecord(start);
	CalcInvertofGlobalSortIdx_2d<<<(M+1024-1)/1024,1024>>>(M,bin_size_x,
		bin_size_y,numbins[0],numbins[1],d_binstartpts,d_sortidx,d_kx,d_ky,
		d_idxnupts);
#ifdef DEBUG
	int *h_idxnupts;
	h_idxnupts = (int*)malloc(M*sizeof(int));
	checkCudaErrors(hipMemcpy(h_idxnupts,d_idxnupts,M*sizeof(int),
				hipMemcpyDeviceToHost));
	for (int i=0; i<M; i++){
		cout <<"[debug ] idx="<< h_idxnupts[i]<<endl;
	}
	free(h_idxnupts);
#endif
	hipEventRecord(start);
	CalcSubProb_2d<<<(M+1024-1)/1024, 1024>>>(d_binsize,d_numsubprob,
			maxsubprobsize,numbins[0]*numbins[1]);
#ifdef DEBUG
	int* h_numsubprob;
	h_numsubprob = (int*) malloc(n*sizeof(int));
	checkCudaErrors(hipMemcpy(h_numsubprob,d_numsubprob,numbins[0]*numbins[1]*
				sizeof(int),hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<
				h_numsubprob[i+j*numbins[0]];
		}
		cout<<endl;
	}
	free(h_numsubprob);
#endif
	// Scanning the same length array, so we don't need calculate 
	// temp_storage_bytes here
	CubDebugExit(hipcub::DeviceScan::InclusiveSum(d_temp_storage, 
				temp_storage_bytes, d_numsubprob, d_subprobstartpts+1, n));
	checkCudaErrors(hipMemset(d_subprobstartpts,0,sizeof(int)));

#ifdef DEBUG
	printf("[debug ] Subproblem start points\n");
	int* h_subprobstartpts;
	h_subprobstartpts = (int*) malloc((n+1)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprobstartpts,d_subprobstartpts,
				(n+1)*sizeof(int),hipMemcpyDeviceToHost));
	for(int j=0; j<numbins[1]; j++){
		cout<<"[debug ] ";
		for(int i=0; i<numbins[0]; i++){
			if(i!=0) cout<<" ";
			cout <<"nsub["<<setw(3)<<i<<","<<setw(3)<<j<<"] = "<<setw(2)<<
				h_subprobstartpts[i+j*numbins[0]];
		}
		cout<<endl;
	}
	printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
	free(h_subprobstartpts);
#endif
	int totalnumsubprob;
	checkCudaErrors(hipMemcpy(&totalnumsubprob,&d_subprobstartpts[n],
				sizeof(int),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMalloc(&d_subprob_to_bin,totalnumsubprob*sizeof(int)));
	MapBintoSubProb_2d<<<(numbins[0]*numbins[1]+1024-1)/1024, 1024>>>(
			d_subprob_to_bin,d_subprobstartpts,d_numsubprob,numbins[0]*numbins[1]);
	assert(d_subprob_to_bin != NULL);
	d_plan->subprob_to_bin = d_subprob_to_bin;
	assert(d_plan->subprob_to_bin != NULL);
	d_plan->totalnumsubprob = totalnumsubprob;
#ifdef DEBUG
	printf("[debug ] Map Subproblem to Bins\n");
	int* h_subprob_to_bin;
	h_subprob_to_bin = (int*) malloc((totalnumsubprob)*sizeof(int));
	checkCudaErrors(hipMemcpy(h_subprob_to_bin,d_subprob_to_bin,
				(totalnumsubprob)*sizeof(int),hipMemcpyDeviceToHost));
	for(int j=0; j<totalnumsubprob; j++){
		cout<<"[debug ] ";
		cout <<"nsub["<<j<<"] = "<<setw(2)<<h_subprob_to_bin[j];
		cout<<endl;
	}
	free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
	hipFree(d_temp_storage);
	return 0;
}

int cuspread2d_subprob(int nf1, int nf2, int M, const cufinufft_opts opts, 
		cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int maxsubprobsize=opts.maxsubprobsize;

	// assume that bin_size_x > ns/2;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int numbins[2];
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;

	int totalnumsubprob=d_plan->totalnumsubprob;
	int *d_subprob_to_bin = d_plan->subprob_to_bin;

	FLT sigma=opts.upsampfac;
	hipEventRecord(start);
	size_t sharedplanorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*
			ceil(ns/2.0))*sizeof(CUCPX);
	if(sharedplanorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	if(opts.Horner){
		for(int t=0; t<d_plan->ntransfcufftplan; t++){
			Spread_2d_Subprob_Horner<<<totalnumsubprob, 256, 
				sharedplanorysize>>>(d_kx, d_ky, d_c+t*M, d_fw+t*nf1*nf2, M, 
				ns, nf1, nf2, sigma, d_binstartpts, d_binsize, bin_size_x, 
				bin_size_y, d_subprob_to_bin, d_subprobstartpts, 
				d_numsubprob, maxsubprobsize,numbins[0], numbins[1], 
				d_idxnupts);
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Subprob_Horner \t%.3g ms\n", 
		milliseconds);
#endif
		}
	}else{
		for(int t=0; t<d_plan->ntransfcufftplan; t++){
			Spread_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
					d_kx, d_ky, d_c+t*M, d_fw+t*nf1*nf2, M, ns, nf1, nf2, 
					es_c, es_beta, sigma,d_binstartpts, d_binsize, bin_size_x, 
					bin_size_y, d_subprob_to_bin, d_subprobstartpts,
					d_numsubprob, maxsubprobsize, numbins[0], numbins[1],
					d_idxnupts);
		}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Spread_2d_Subprob\t\t%.3g ms\n", milliseconds);
#endif
	}
	return 0;
}
