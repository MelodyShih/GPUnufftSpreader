#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "../spreadinterp.h"
#include "../memtransfer.h"

using namespace std;

// This function includes device memory allocation, transfer, free
int cufinufft_interp2d(int ms, int mt, int nf1, int nf2, CPX* h_fw, int M, FLT *h_kx,
		FLT *h_ky, CPX *h_c, cufinufft_opts &opts, cufinufft_plan* d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;

	d_plan->ms = ms;
	d_plan->mt = mt;
	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->M = M;
	d_plan->ntransfcufftplan = 1;

	hipEventRecord(start);
	ier = allocgpumemory2d(opts, d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Allocate GPU memory\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	hipMemcpy2D(d_plan->fw,d_plan->fw_width*sizeof(CUCPX),h_fw,nf1*sizeof(CUCPX),
			nf1*sizeof(CUCPX),nf2,hipMemcpyHostToDevice);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory HtoD\t %.3g ms\n", milliseconds);
#endif
	if(opts.method == 5){
		ier = cuspread2d_subprob_prop(nf1,nf2,M,opts,d_plan);
		if(ier != 0 ){
			printf("error: cuspread2d_subprob_prop, method(%d)\n", opts.method);
			return 0;
		}
	}
	hipEventRecord(start);
	ier = cuinterp2d(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Interp (%d)\t\t %.3g ms\n", opts.method, milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(h_c,d_plan->c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory DtoH\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	freegpumemory2d(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
	return ier;
}

// a wrapper of different methods of spreader
int cuinterp2d(cufinufft_opts &opts, cufinufft_plan* d_plan)
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int fw_width = d_plan->fw_width;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch(opts.method)
	{
		case 1:
			{
				hipEventRecord(start);
				ier = cuinterp2d_idriven(nf1, nf2, fw_width, M, opts, d_plan);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_idriven"<<endl;
					return 1;
				}
			}
			break;
		case 5:
			{
				hipEventRecord(start);
				ier = cuinterp2d_subprob(nf1, nf2, fw_width, M, opts, d_plan);
				if(ier != 0 ){
					cout<<"error: cnufftspread2d_gpu_hybrid"<<endl;
					return 1;
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 1 or 5"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	float milliseconds;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Interp " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}

int cuinterp2d_idriven(int nf1, int nf2, int fw_width, int M, const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;
	hipEventRecord(start);

	for(int t=0; t<d_plan->ntransfcufftplan; t++){
		Interp_2d_Idriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c+t*M, d_fw+t*nf1*nf2, 
				M, ns, nf1, nf2, es_c, es_beta, 
				fw_width);
	}
#ifdef SPREADTIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_2d_Idriven \t%.3g ms\n", milliseconds);
#endif
	return 0;
}

int cuinterp2d_subprob(int nf1, int nf2, int fw_width, int M, const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int maxsubprobsize=opts.maxsubprobsize;

	// assume that bin_size_x > ns/2;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int numbins[2];
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"]"<<endl;
#endif

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;
	int *d_subprob_to_bin = d_plan->subprob_to_bin;
	int totalnumsubprob=d_plan->totalnumsubprob;

	FLT sigma=opts.upsampfac;
	hipEventRecord(start);
	size_t sharedplanorysize = (bin_size_x+2*ceil(ns/2.0))*(bin_size_y+2*ceil(ns/2.0))*sizeof(CUCPX);
	if(sharedplanorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	for(int t=0; t<d_plan->ntransfcufftplan; t++){
		Interp_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
				d_kx, d_ky, d_c+t*M,
				d_fw+t*nf1*nf2, M, ns, nf1, nf2,
				es_c, es_beta, sigma, fw_width,
				d_binstartpts, d_binsize,
				bin_size_x, bin_size_y,
				d_subprob_to_bin, d_subprobstartpts,
				d_numsubprob, maxsubprobsize,
				numbins[0], numbins[1], d_idxnupts);
	}
#ifdef SPREADTIME
 	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] \tKernel Interp_2d_Subprob \t\t%.3g ms\n", milliseconds);
#endif
	return 0;
}
