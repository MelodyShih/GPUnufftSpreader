#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "deconvolve.h"

using namespace std;

__global__
void Deconvolve_2d(int ms, int mt, int nf1, int nf2, int fw_width, CUCPX* fw, CUCPX *fk, 
                   FLT *fwkerhalf1, FLT *fwkerhalf2)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<ms*mt; i+=blockDim.x*gridDim.x){
		int k1 = i % ms;
		int k2 = i / ms;
		int outidx = k1 + k2*ms;
		int w1 = k1-ms/2 > 0 ? k1-ms/2 : nf1+k1-ms/2;
		int w2 = k2-mt/2 > 0 ? k2-mt/2 : nf2+k2-mt/2;
		int inidx = w1 + w2*fw_width;

		FLT kervalue = fwkerhalf1[abs(k1-ms/2)]*fwkerhalf2[abs(k2-mt/2)];
		fk[outidx].x = fw[inidx].x/kervalue;
		fk[outidx].y = fw[inidx].y/kervalue;
		//fk[outidx].x = kervalue;
		//fk[outidx].y = kervalue;
	}
}

int cudeconvolve2d(spread_opts opts, cufinufft_plan *d_plan)
// ms = N1
// mt = N2
{
	int ms=d_plan->ms;
	int mt=d_plan->mt;
	int nf1=d_plan->nf1;
	int nf2=d_plan->nf2;
	int fw_width=d_plan->fw_width;
	int nmodes=ms*mt;
	Deconvolve_2d<<<(nmodes+256-1)/256, 256>>>(ms, mt, nf1, nf2, fw_width, d_plan->fw, d_plan->fk,
						   d_plan->fwkerhalf1, d_plan->fwkerhalf2);
	return 0;
}

