#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

#include <hip/hip_complex.h>
#include "deconvolve.h"

using namespace std;

/* Kernel for copying fw to fk with amplication by prefac/ker */
// Note: assume modeord=0: CMCL-compatible mode ordering in fk (from -N/2 up 
// to N/2-1)
__global__
void Deconvolve_2d(int ms, int mt, int nf1, int nf2, CUCPX* fw, CUCPX *fk, 
		FLT *fwkerhalf1, FLT *fwkerhalf2)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<ms*mt; i+=blockDim.x*gridDim.x){
		int k1 = i % ms;
		int k2 = i / ms;
		int outidx = k1 + k2*ms;
		int w1 = k1-ms/2 >= 0 ? k1-ms/2 : nf1+k1-ms/2;
		int w2 = k2-mt/2 >= 0 ? k2-mt/2 : nf2+k2-mt/2;
		int inidx = w1 + w2*nf1;

		FLT kervalue = fwkerhalf1[abs(k1-ms/2)]*fwkerhalf2[abs(k2-mt/2)];
		fk[outidx].x = fw[inidx].x/kervalue;
		fk[outidx].y = fw[inidx].y/kervalue;
	}
}

/* Kernel for copying fk to fw with same amplication */
__global__
void Amplify_2d(int ms, int mt, int nf1, int nf2, CUCPX* fw, CUCPX *fk, 
		FLT *fwkerhalf1, FLT *fwkerhalf2)
{
	for(int i=blockDim.x*blockIdx.x+threadIdx.x; i<ms*mt; i+=blockDim.x*gridDim.x){
		int k1 = i % ms;
		int k2 = i / ms;
		int inidx = k1 + k2*ms;
		int w1 = k1-ms/2 >= 0 ? k1-ms/2 : nf1+k1-ms/2;
		int w2 = k2-mt/2 >= 0 ? k2-mt/2 : nf2+k2-mt/2;
		int outidx = w1 + w2*nf1;

		FLT kervalue = fwkerhalf1[abs(k1-ms/2)]*fwkerhalf2[abs(k2-mt/2)];
		fw[outidx].x = fk[inidx].x/kervalue;
		fw[outidx].y = fk[inidx].y/kervalue;
	}
}

/* CPU wrapper for deconvolution & amplication */
int cudeconvolve2d(cufinufft_plan *d_plan)
{
	int ms=d_plan->ms;
	int mt=d_plan->mt;
	int nf1=d_plan->nf1;
	int nf2=d_plan->nf2;
	int nmodes=ms*mt;
	int ntransfcufftplan=d_plan->ntransfcufftplan;

	if(d_plan->spopts.spread_direction == 1){
		for(int t=0; t<ntransfcufftplan; t++){
			Deconvolve_2d<<<(nmodes+256-1)/256, 256>>>(ms, mt, nf1, nf2, 
				d_plan->fw+t*nf1*nf2,d_plan->fk+t*nmodes,d_plan->fwkerhalf1, 
				d_plan->fwkerhalf2);
		}
	}else{
		checkCudaErrors(hipMemset(d_plan->fw,0,ntransfcufftplan*nf1*nf2*
			sizeof(CUCPX)));
		for(int t=0; t<ntransfcufftplan; t++){
			Amplify_2d<<<(nmodes+256-1)/256, 256>>>(ms, 
				mt, nf1, nf2, d_plan->fw+t*nf1*nf2, d_plan->fk+t*nmodes,
				d_plan->fwkerhalf1, d_plan->fwkerhalf2);
#ifdef DEBUG
			CPX* h_fw;
			h_fw = (CPX*) malloc(nf1*nf2*sizeof(CPX));
			checkCudaErrors(hipMemcpy2D(h_fw,nf1*sizeof(CUCPX),d_plan->fw,nf1*sizeof(CUCPX),
						nf1*sizeof(CUCPX),nf2,hipMemcpyDeviceToHost));
			for(int j=0; j<nf2; j++){
				for(int i=0; i<nf1; i++){
					printf("(%g,%g)",h_fw[i+j*nf1].real(),h_fw[i+j*nf1].imag());
				}
				printf("\n");
			}
			free(h_fw);
#endif
		}
	}
	return 0;
}

