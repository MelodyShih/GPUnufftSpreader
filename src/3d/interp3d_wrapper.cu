#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>

// try another library cub
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>

#include <hip/hip_complex.h>
#include "../spreadinterp.h"
#include "../memtransfer.h"
#include "../profile.h"

using namespace std;

// This function includes device memory allocation, transfer, free
int cufinufft_interp3d(int ms, int mt, int mu, int nf1, int nf2, int nf3, 
	CPX* h_fw, int M, FLT *h_kx, FLT *h_ky, FLT *h_kz, CPX *h_c, 
	cufinufft_opts &opts, cufinufft_plan* d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;

	d_plan->ms = ms;
	d_plan->mt = mt;
	d_plan->mu = mu;
	d_plan->nf1 = nf1;
	d_plan->nf2 = nf2;
	d_plan->nf3 = nf3;
	d_plan->M = M;
	d_plan->ntransfcufftplan = 1;

	hipEventRecord(start);
	ier = allocgpumemory3d(opts, d_plan);
#ifdef TIME
	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Allocate GPU memory\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(d_plan->kx,h_kx,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->ky,h_ky,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->kz,h_kz,M*sizeof(FLT),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_plan->fw,h_fw,nf1*nf2*nf3*sizeof(CUCPX),
		hipMemcpyHostToDevice));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory HtoD\t %.3g ms\n", milliseconds);
#endif
	if(opts.method == 5){
		ier = cuspread3d_subprob_prop(nf1,nf2,nf3,M,opts,d_plan);
		if(ier != 0 ){
			printf("error: cuspread3d_subprob_prop, method(%d)\n", opts.method);
			return 0;
		}
	}
	hipEventRecord(start);
	ier = cuinterp3d(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Interp (%d)\t\t %.3g ms\n", opts.method, milliseconds);
#endif
	hipEventRecord(start);
	checkCudaErrors(hipMemcpy(h_c,d_plan->c,M*sizeof(CUCPX),hipMemcpyDeviceToHost));
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Copy memory DtoH\t %.3g ms\n", milliseconds);
#endif
	hipEventRecord(start);
	freegpumemory3d(opts, d_plan);
#ifdef TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
	return ier;
}

// a wrapper of different methods of spreader
int cuinterp3d(cufinufft_opts &opts, cufinufft_plan* d_plan)
{
	int nf1 = d_plan->nf1;
	int nf2 = d_plan->nf2;
	int nf3 = d_plan->nf3;
	int M = d_plan->M;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int ier;
	switch(opts.method)
	{
		case 4:
			{
				hipEventRecord(start);
				{
					PROFILE_CUDA_GROUP("Interpolation", 6);
					ier = cuinterp3d_idriven(nf1, nf2, nf3, M, opts, d_plan);
					if(ier != 0 ){
						cout<<"error: cnufftspread3d_gpu_idriven"<<endl;
						return 1;
					}
				}
			}
			break;
		case 5:
			{
				hipEventRecord(start);
				{
					PROFILE_CUDA_GROUP("Interpolation", 6);
					ier = cuinterp3d_subprob(nf1, nf2, nf3, M, opts, d_plan);
					if(ier != 0 ){
						cout<<"error: cnufftspread3d_gpu_subprob"<<endl;
						return 1;
					}
				}
			}
			break;
		default:
			cout<<"error: incorrect method, should be 4"<<endl;
			return 2;
	}
#ifdef SPREADTIME
	float milliseconds;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"[time  ]"<< " Interp " << milliseconds <<" ms"<<endl;
#endif
	return ier;
}

int cuinterp3d_idriven(int nf1, int nf2, int nf3, int M, 
	const cufinufft_opts opts, cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	FLT sigma=opts.upsampfac;

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	FLT* d_kz = d_plan->kz;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	threadsPerBlock.x = 16;
	threadsPerBlock.y = 1;
	blocks.x = (M + threadsPerBlock.x - 1)/threadsPerBlock.x;
	blocks.y = 1;

	hipEventRecord(start);
	if(opts.Horner){
#if 0
		hipStream_t *streams = d_plan->streams;
		int nstreams = d_plan->nstreams;
		for(int t=0; t<d_plan->ntransfcufftplan; t++){
			Interp_3d_Idriven_Horner<<<blocks, threadsPerBlock, 0, 
				streams[t%nstreams]>>>(d_kx, d_ky, d_kz, d_c+t*M, 
				d_fw+t*nf1*nf2*nf3, M, ns, nf1, nf2, nf3, sigma);
		}
#else 
		for(int t=0; t<d_plan->ntransfcufftplan; t++){
			Interp_3d_Idriven_Horner<<<blocks, threadsPerBlock, 0, 
				0>>>(d_kx, d_ky, d_kz, d_c+t*M, 
				d_fw+t*nf1*nf2*nf3, M, ns, nf1, nf2, nf3, sigma);
		}
#endif
#ifdef SPREADTIME
			float milliseconds = 0;
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milliseconds, start, stop);
			printf("[time  ] \tKernel Interp_3d_Idriven_Horner \t%.3g ms\n", milliseconds);
#endif
	}else{
#if 0
		hipStream_t *streams = d_plan->streams;
		int nstreams = d_plan->nstreams;
		for(int t=0; t<d_plan->ntransfcufftplan; t++){
			Interp_3d_Idriven<<<blocks, threadsPerBlock, 0, streams[t%nstreams]
				>>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, M, ns, 
				nf1, nf2, nf3,es_c, es_beta);
		}
#else
		for(int t=0; t<d_plan->ntransfcufftplan; t++){
			Interp_3d_Idriven<<<blocks, threadsPerBlock, 0, 0 
				>>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, M, ns, 
				nf1, nf2, nf3,es_c, es_beta);
		}
#endif
#ifdef SPREADTIME
			float milliseconds = 0;
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milliseconds, start, stop);
			printf("[time  ] \tKernel Interp_3d_Idriven \t%.3g ms\n", milliseconds);
#endif
	}
	return 0;
}

int cuinterp3d_subprob(int nf1, int nf2, int nf3, int M, const cufinufft_opts opts, 
	cufinufft_plan *d_plan)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threadsPerBlock;
	dim3 blocks;

	int ns=opts.nspread;   // psi's support in terms of number of cells
	FLT es_c=opts.ES_c;
	FLT es_beta=opts.ES_beta;
	int maxsubprobsize=opts.maxsubprobsize;

	// assume that bin_size_x > ns/2;
	int bin_size_x=opts.bin_size_x;
	int bin_size_y=opts.bin_size_y;
	int bin_size_z=opts.bin_size_z;
	int numbins[3];
	numbins[0] = ceil((FLT) nf1/bin_size_x);
	numbins[1] = ceil((FLT) nf2/bin_size_y);
	numbins[2] = ceil((FLT) nf3/bin_size_z);
#ifdef INFO
	cout<<"[info  ] Dividing the uniform grids to bin size["
		<<opts.bin_size_x<<"x"<<opts.bin_size_y<<"x"<<opts.bin_size_z<<"]"<<endl;
	cout<<"[info  ] numbins = ["<<numbins[0]<<"x"<<numbins[1]<<"x"<<numbins[2]
	<<"]"<<endl;
#endif

	FLT* d_kx = d_plan->kx;
	FLT* d_ky = d_plan->ky;
	FLT* d_kz = d_plan->kz;
	CUCPX* d_c = d_plan->c;
	CUCPX* d_fw = d_plan->fw;

	int *d_binsize = d_plan->binsize;
	int *d_binstartpts = d_plan->binstartpts;
	int *d_numsubprob = d_plan->numsubprob;
	int *d_subprobstartpts = d_plan->subprobstartpts;
	int *d_idxnupts = d_plan->idxnupts;
	int *d_subprob_to_bin = d_plan->subprob_to_bin;
	int totalnumsubprob=d_plan->totalnumsubprob;

	FLT sigma=opts.upsampfac;
	hipEventRecord(start);
	size_t sharedplanorysize = (bin_size_x+2*ceil(ns/2.0))*
		(bin_size_y+2*ceil(ns/2.0))*(bin_size_z+2*ceil(ns/2.0))*sizeof(CUCPX);
	if(sharedplanorysize > 49152){
		cout<<"error: not enough shared memory"<<endl;
		return 1;
	}

	for(int t=0; t<d_plan->ntransfcufftplan; t++){
		Interp_3d_Subprob_Horner<<<totalnumsubprob, 256,
			sharedplanorysize>>>(d_kx, d_ky, d_kz, d_c+t*M, d_fw+t*nf1*nf2*nf3, 
			M, ns, nf1, nf2, nf3, sigma, d_binstartpts, d_binsize, bin_size_x,
			bin_size_y, bin_size_z, d_subprob_to_bin, d_subprobstartpts,
			d_numsubprob, maxsubprobsize,numbins[0], numbins[1], numbins[2],
			d_idxnupts);
#ifdef SPREADTIME
			float milliseconds = 0;
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milliseconds, start, stop);
			printf("[time  ] \tKernel Interp_3d_Subprob_Horner \t%.3g ms\n", milliseconds);
#endif
	}
	return 0;
}
